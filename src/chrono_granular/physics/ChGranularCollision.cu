#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2018 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// Contains some collision helper functions for chrono_granular, lifted from ChNarrowphaseR
//
// =============================================================================
// Authors: Dan Negrut, Conlain Kelly
// =============================================================================
#include "chrono_granular/ChGranularDefines.h"
#include "chrono_granular/physics/ChGranularCollision.cuh"
#include "chrono_granular/utils/ChCudaMathUtils.cuh"


/// This utility function takes the location 'P' and snaps it to the closest
/// point on the triangular face with given vertices (A, B, and C). The result
/// is returned in 'res'. Both 'P' and 'res' are assumed to be specified in
/// the same frame as the face vertices. This function returns 'true' if the
/// result is on an edge of this face and 'false' if the result is inside the
/// triangle.
/// Code from Ericson, "real-time collision detection", 2005, pp. 141
__device__ bool snap_to_face(const float3& A, const float3& B, const float3& C, const float3& P, float3& res) {
    float3 AB = B - A;
    float3 AC = C - A;

    // Check if P in vertex region outside A
    float3 AP = P - A;
    float d1 = Dot(AB, AP);
    float d2 = Dot(AC, AP);
    if (d1 <= 0 && d2 <= 0) {
        res = A;  // barycentric coordinates (1,0,0)
        return true;
    }

    // Check if P in vertex region outside B
    float3 BP = P - B;
    float d3 = Dot(AB, BP);
    float d4 = Dot(AC, BP);
    if (d3 >= 0 && d4 <= d3) {
        res = B;  // barycentric coordinates (0,1,0)
        return true;
    }

    // Check if P in edge region of AB
    float vc = d1 * d4 - d3 * d2;
    if (vc <= 0 && d1 >= 0 && d3 <= 0) {
        // Return projection of P onto AB
        float v = d1 / (d1 - d3);
        res = A + v * AB;  // barycentric coordinates (1-v,v,0)
        return true;
    }

    // Check if P in vertex region outside C
    float3 CP = P - C;
    float d5 = Dot(AB, CP);
    float d6 = Dot(AC, CP);
    if (d6 >= 0 && d5 <= d6) {
        res = C;  // barycentric coordinates (0,0,1)
        return true;
    }

    // Check if P in edge region of AC
    float vb = d5 * d2 - d1 * d6;
    if (vb <= 0 && d2 >= 0 && d6 <= 0) {
        // Return projection of P onto AC
        float w = d2 / (d2 - d6);
        res = A + w * AC;  // barycentric coordinates (1-w,0,w)
        return true;
    }

    // Check if P in edge region of BC
    float va = d3 * d6 - d5 * d4;
    if (va <= 0 && (d4 - d3) >= 0 && (d5 - d6) >= 0) {
        // Return projection of P onto BC
        float w = (d4 - d3) / ((d4 - d3) + (d5 - d6));
        res = B + w * (C - B);  // barycentric coordinates (0,1-w,w)
        return true;
    }

    // P inside face region. Return projection of P onto face
    // barycentric coordinates (u,v,w)
    float denom = 1 / (va + vb + vc);
    float v = vb * denom;
    float w = vc * denom;
    res = A + v * AB + w * AC;  // = u*A + v*B + w*C  where  (u = 1 - v - w)
    return false;
}

// =============================================================================
//              FACE - SPHERE
// Face-sphere narrow phase collision detection.
// In: triangular face defined by points A1, B1, C1
//     sphere sphere centered at pos2 and with radius2
__device__ bool face_sphere(const float3& A1,
                            const float3& B1,
                            const float3& C1,
                            const float3& pos2,
                            const float& radius2,
                            const float& separation,
                            float3& norm,
                            float& depth,
                            float3& pt1,
                            float3& pt2,
                            float& eff_radius) {
    float radius2_s = radius2 + separation;

    // Calculate face normal.
    float3 nrm1 = face_normal(A1, B1, C1);

    // Calculate signed height of sphere center above face plane. If the
    // height is larger than the sphere radius plus the separation value
    // or if the sphere center is below the plane, there is no contact.
    float h = Dot(pos2 - A1, nrm1);

    if (h >= radius2_s || h <= 0.f)
        return false;

    // Find the closest point on the face to the sphere center and determine
    // whether or not this location is inside the face or on an edge.
    float3 faceLoc;

    if (snap_to_face(A1, B1, C1, pos2, faceLoc)) {
        // Closest face feature is an edge. If the distance between the sphere
        // center and the closest point is more than the radius plus the
        // separation value, then there is no contact. Also, ignore contact if
        // the sphere center (almost) coincides with the closest point, in
        // which case we couldn't decide on the proper contact direction.
        float3 delta = pos2 - faceLoc;
        float dist2 = Dot(delta, delta);

        if (dist2 >= radius2_s * radius2_s || dist2 <= 1e-12f)
            return false;

        float dist = sqrt(dist2);
        norm = delta / dist;
        depth = dist - radius2;
        eff_radius = radius2 * EDGE_RADIUS / (radius2 + EDGE_RADIUS);
    } else {
        // Closest point on face is inside the face.
        norm = nrm1;
        depth = h - radius2;
        eff_radius = radius2;
    }

    pt1 = faceLoc;
    pt2 = pos2 - norm * radius2;

    return true;
}