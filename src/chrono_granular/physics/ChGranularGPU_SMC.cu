#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2018 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Dan Negrut, Conlain Kelly, Nic Olsen
// =============================================================================

#include "../chrono_granular/physics/ChGranularGPU_SMC.cuh"

namespace chrono {
namespace granular {

__host__ double ChSystemGranular::get_max_z() const {
    int* max_z_d;
    int max_z_h;
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    gpuErrchk(hipMalloc(&max_z_d, sizeof(int)));

    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, pos_Z.data(), max_z_d, nDEs);
    gpuErrchk(hipDeviceSynchronize());

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run max-reduction
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, pos_Z.data(), max_z_d, nDEs);
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(&max_z_h, max_z_d, sizeof(int), hipMemcpyDeviceToHost));

    double max_z_UU = max_z_h * gran_params->LENGTH_UNIT;
    gpuErrchk(hipFree(max_z_d));
    gpuErrchk(hipDeviceSynchronize());

    return max_z_UU;
}

/// Copy constant sphere data to device, this should run at start
__host__ void ChSystemGranularMonodisperse_SMC::copy_const_data_to_device() {
    // Copy quantities expressed in SU units for the SD dimensions to device
    gran_params->SD_size_X_SU = SD_size_X_SU;
    gran_params->SD_size_Y_SU = SD_size_Y_SU;
    gran_params->SD_size_Z_SU = SD_size_Z_SU;
    // Copy global BD size in multiples of SDs to device
    gran_params->nSDs_X = nSDs_X;
    gran_params->nSDs_Y = nSDs_Y;
    gran_params->nSDs_Z = nSDs_Z;

    gran_params->gravAcc_X_SU = gravity_X_SU;
    gran_params->gravAcc_Y_SU = gravity_Y_SU;
    gran_params->gravAcc_Z_SU = gravity_Z_SU;
    gran_params->gravMag_SU =
        std::sqrt(gravity_X_SU * gravity_X_SU + gravity_Y_SU * gravity_Y_SU + gravity_Z_SU * gravity_Z_SU);

    gran_params->sphereRadius_SU = sphereRadius_SU;

    gran_params->Kn_s2s_SU = K_n_s2s_SU;
    gran_params->Kn_s2w_SU = K_n_s2w_SU;
    gran_params->Gamma_n_s2s_SU = Gamma_n_s2s_SU;
    gran_params->Gamma_n_s2w_SU = Gamma_n_s2w_SU;

    gran_params->K_t_s2s_SU = K_t_s2s_SU;
    gran_params->K_t_s2w_SU = K_t_s2w_SU;
    gran_params->Gamma_t_s2s_SU = Gamma_t_s2s_SU;
    gran_params->Gamma_t_s2w_SU = Gamma_t_s2w_SU;

    gran_params->cohesion_ratio = cohesion_over_gravity;

    gran_params->integrator_type = time_integrator;
    gran_params->contact_model = contact_model;
    gran_params->friction_mode = fric_mode;
}

/// Similar to the copy_const_data_to_device, but saves us a big copy
/// This can run at every timestep to allow a moving BD
__host__ void ChSystemGranularMonodisperse_SMC::copyBD_Frame_to_device() {
    // Unified memory does all the work here
    gran_params->BD_frame_X = BD_frame_X;
    gran_params->BD_frame_Y = BD_frame_Y;
    gran_params->BD_frame_Z = BD_frame_Z;
    gran_params->BD_frame_X_dot = BD_frame_X_dot;
    gran_params->BD_frame_Y_dot = BD_frame_Y_dot;
    gran_params->BD_frame_Z_dot = BD_frame_Z_dot;
}

// Check number of spheres in each SD and dump relevant info to file
void ChSystemGranularMonodisperse_SMC::checkSDCounts(std::string ofile, bool write_out = false, bool verbose = false) {
    // Count of DEs in each SD
    unsigned int* sdvals = SD_NumOf_DEs_Touching.data();
    // DEs that are in each SD
    unsigned int* sdSpheres = DEs_in_SD_composite.data();
    // # times each DE appears in some SD
    unsigned int* deCounts = new unsigned int[nDEs];

    // could use memset instead, just need to zero these out
    for (unsigned int i = 0; i < nDEs; i++) {
        deCounts[i] = 0;
    }

    unsigned int max_count = 0;
    unsigned int sum = 0;
    for (unsigned int i = 0; i < nSDs; i++) {
        // printf("count is %u for SD sd %u \n", sdvals[i], i);
        sum += sdvals[i];
        if (sdvals[i] > max_count)
            max_count = sdvals[i];
    }
    // safety checks, if these fail we were probably about to crash
    assert(sum < MAX_COUNT_OF_DEs_PER_SD * nSDs);
    assert(max_count < MAX_COUNT_OF_DEs_PER_SD);
    if (verbose) {
        printf("max DEs per SD is %u\n", max_count);
        printf("total sd/de overlaps is %u\n", sum);
        printf("theoretical total is %u\n", MAX_COUNT_OF_DEs_PER_SD * nSDs);
    }
    // Copy over occurences in SDs
    for (unsigned int i = 0; i < MAX_COUNT_OF_DEs_PER_SD * nSDs; i++) {
        // printf("de id is %d, i is %u\n", sdSpheres[i], i);
        // Check if invalid sphere
        if (sdSpheres[i] == NULL_GRANULAR_ID) {
            // printf("invalid sphere in sd");
        } else {
            assert(sdSpheres[i] < nDEs);
            deCounts[sdSpheres[i]]++;
        }
    }
    if (write_out) {
        writeFile(ofile, deCounts);
    }
    delete[] deCounts;
}
// This can belong to the superclass but does reference deCounts which may not be a thing when DVI rolls around
void ChSystemGranularMonodisperse_SMC::writeFile(std::string ofile, unsigned int* deCounts) {
    // unnecessary if called by checkSDCounts()
    // The file writes are a pretty big slowdown in CSV mode
    if (file_write_mode == GRAN_OUTPUT_MODE::BINARY) {
        // Write the data as binary to a file, requires later postprocessing that can be done in parallel, this is a
        // much faster write due to no formatting
        std::ofstream ptFile(ofile + ".raw", std::ios::out | std::ios::binary);

        for (unsigned int n = 0; n < nDEs; n++) {
            float absv = sqrt(pos_X_dt.at(n) * pos_X_dt.at(n) + pos_Y_dt.at(n) * pos_Y_dt.at(n) +
                              pos_Z_dt.at(n) * pos_Z_dt.at(n));

            ptFile.write((const char*)&pos_X.at(n), sizeof(int));
            ptFile.write((const char*)&pos_Y.at(n), sizeof(int));
            ptFile.write((const char*)&pos_Z.at(n), sizeof(int));
            ptFile.write((const char*)&pos_X_dt.at(n), sizeof(float));
            ptFile.write((const char*)&pos_Y_dt.at(n), sizeof(float));
            ptFile.write((const char*)&pos_Z_dt.at(n), sizeof(float));
            ptFile.write((const char*)&absv, sizeof(float));
            ptFile.write((const char*)&deCounts[n], sizeof(int));
        }
    } else if (file_write_mode == GRAN_OUTPUT_MODE::CSV) {
        // CSV is much slower but requires less postprocessing
        std::ofstream ptFile(ofile + ".csv", std::ios::out);

        // Dump to a stream, write to file only at end
        std::ostringstream outstrstream;
        outstrstream << "x,y,z,vx,vy,vz,absv,nTouched\n";

        for (unsigned int n = 0; n < nDEs; n++) {
            float absv = sqrt(pos_X_dt.at(n) * pos_X_dt.at(n) + pos_Y_dt.at(n) * pos_Y_dt.at(n) +
                              pos_Z_dt.at(n) * pos_Z_dt.at(n));
            outstrstream << pos_X.at(n) << "," << pos_Y.at(n) << "," << pos_Z.at(n) << "," << pos_X_dt.at(n) << ","
                         << pos_Y_dt.at(n) << "," << pos_Z_dt.at(n) << "," << absv << "," << deCounts[n] << "\n";
        }

        ptFile << outstrstream.str();
    } else if (file_write_mode == GRAN_OUTPUT_MODE::NONE) {
        // Do nothing, only here for symmetry
    }
}

// This can belong to the superclass but does reference deCounts which may not be a thing when DVI rolls around
void ChSystemGranularMonodisperse_SMC::writeFileUU(std::string ofile) {
    // The file writes are a pretty big slowdown in CSV mode
    if (file_write_mode == GRAN_OUTPUT_MODE::BINARY) {
        // TODO implement this
        // Write the data as binary to a file, requires later postprocessing that can be done in parallel, this is a
        // much faster write due to no formatting
        // std::ofstream ptFile(ofile + ".raw", std::ios::out | std::ios::binary);
        //
        // for (unsigned int n = 0; n < nDEs; n++) {
        //     float absv = sqrt(pos_X_dt.at(n) * pos_X_dt.at(n) + pos_Y_dt.at(n) * pos_Y_dt.at(n) +
        //                       pos_Z_dt.at(n) * pos_Z_dt.at(n));
        //
        //     ptFile.write((const char*)&pos_X.at(n), sizeof(int));
        //     ptFile.write((const char*)&pos_Y.at(n), sizeof(int));
        //     ptFile.write((const char*)&pos_Z.at(n), sizeof(int));
        //     ptFile.write((const char*)&pos_X_dt.at(n), sizeof(float));
        //     ptFile.write((const char*)&pos_Y_dt.at(n), sizeof(float));
        //     ptFile.write((const char*)&pos_Z_dt.at(n), sizeof(float));
        //     ptFile.write((const char*)&absv, sizeof(float));
        //     ptFile.write((const char*)&deCounts[n], sizeof(int));
        // }
    } else if (file_write_mode == GRAN_OUTPUT_MODE::CSV) {
        // CSV is much slower but requires less postprocessing
        std::ofstream ptFile(ofile + ".csv", std::ios::out);

        // Dump to a stream, write to file only at end
        std::ostringstream outstrstream;
        outstrstream << "x,y,z,USU,wx,wy,wz\n";

        for (unsigned int n = 0; n < nDEs; n++) {
            // TODO convert absv into UU
            float absv = sqrt(pos_X_dt.at(n) * pos_X_dt.at(n) + pos_Y_dt.at(n) * pos_Y_dt.at(n) +
                              pos_Z_dt.at(n) * pos_Z_dt.at(n));

            outstrstream << pos_X.at(n) * gran_params->LENGTH_UNIT << "," << pos_Y.at(n) * gran_params->LENGTH_UNIT
                         << "," << pos_Z.at(n) * gran_params->LENGTH_UNIT << "," << absv;

            if (fric_mode != GRAN_FRICTION_MODE::FRICTIONLESS) {
                outstrstream << "," << omega_X.at(n) << "," << omega_Y.at(n) << "," << omega_Z.at(n);
            }
            outstrstream << "\n";
        }

        ptFile << outstrstream.str();
    } else if (file_write_mode == GRAN_OUTPUT_MODE::NONE) {
        // Do nothing, only here for symmetry
    }
}

// Reset broadphase data structures
void ChSystemGranularMonodisperse_SMC::resetBroadphaseInformation() {
    // Set all the offsets to zero
    gpuErrchk(hipMemset(SD_NumOf_DEs_Touching.data(), 0, nSDs * sizeof(unsigned int)));
    // For each SD, all the spheres touching that SD should have their ID be NULL_GRANULAR_ID
    gpuErrchk(hipMemset(DEs_in_SD_composite.data(), NULL_GRANULAR_ID,
                         MAX_COUNT_OF_DEs_PER_SD * nSDs * sizeof(unsigned int)));
}
// Reset sphere-sphere force data structures
void ChSystemGranularMonodisperse_SMC::resetSphereForces() {
    // cache past force data
    if (time_integrator == GRAN_TIME_INTEGRATOR::CHUNG) {
        gpuErrchk(hipMemcpy(sphere_force_X_old.data(), sphere_force_X.data(), nDEs * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(sphere_force_Y_old.data(), sphere_force_Y.data(), nDEs * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(sphere_force_Z_old.data(), sphere_force_Z.data(), nDEs * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipDeviceSynchronize());
    }
    // reset forces to zero
    gpuErrchk(hipMemset(sphere_force_X.data(), 0, nDEs * sizeof(float)));
    gpuErrchk(hipMemset(sphere_force_Y.data(), 0, nDEs * sizeof(float)));
    gpuErrchk(hipMemset(sphere_force_Z.data(), 0, nDEs * sizeof(float)));

    // reset torques to zero, if applicable
    if (fric_mode != FRICTIONLESS) {
        gpuErrchk(hipMemset(sphere_torque_X.data(), 0, nDEs * sizeof(float)));
        gpuErrchk(hipMemset(sphere_torque_Y.data(), 0, nDEs * sizeof(float)));
        gpuErrchk(hipMemset(sphere_torque_Z.data(), 0, nDEs * sizeof(float)));
    }
}

void ChSystemGranularMonodisperse_SMC::updateBDPosition(const float stepSize_SU) {
    // Frequency of oscillation
    float frame_X_old = BD_frame_X;
    float frame_Y_old = BD_frame_Y;
    float frame_Z_old = BD_frame_Z;
    // Put the bottom-left corner of box wherever the user told us to
    BD_frame_X = (box_size_X * (BDPositionFunctionX(elapsedSimTime))) / gran_params->LENGTH_UNIT;
    BD_frame_Y = (box_size_Y * (BDPositionFunctionY(elapsedSimTime))) / gran_params->LENGTH_UNIT;
    BD_frame_Z = (box_size_Z * (BDPositionFunctionZ(elapsedSimTime))) / gran_params->LENGTH_UNIT;

    BD_frame_X_dot = (BD_frame_X - frame_X_old) / stepSize_SU;
    BD_frame_Y_dot = (BD_frame_Y - frame_Y_old) / stepSize_SU;
    BD_frame_Z_dot = (BD_frame_Z - frame_Z_old) / stepSize_SU;

    copyBD_Frame_to_device();
}

// All the information a moving sphere needs
typedef struct {
    int pos_X;
    int pos_Y;
    int pos_Z;
    float pos_X_dt;
    float pos_Y_dt;
    float pos_Z_dt;
} sphere_data_struct;

template <unsigned int CUB_THREADS>
__global__ void owner_prepack(int* d_sphere_pos_X,
                              int* d_sphere_pos_Y,
                              int* d_sphere_pos_Z,
                              float* d_sphere_pos_X_dt,
                              float* d_sphere_pos_Y_dt,
                              float* d_sphere_pos_Z_dt,
                              unsigned int nSpheres,
                              unsigned int* owners,
                              sphere_data_struct* sphere_info,
                              ParamsPtr gran_params) {
    // Figure out what sphereID this thread will handle. We work with a 1D block structure and a 1D grid structure
    unsigned int mySphereID = threadIdx.x + blockIdx.x * blockDim.x;
    // Only do this for valid spheres
    if (mySphereID >= nSpheres) {
        return;
    }
    // Find this SD's owner
    owners[mySphereID] = SDTripletID(
        pointSDTriplet(d_sphere_pos_X[mySphereID], d_sphere_pos_Y[mySphereID], d_sphere_pos_Z[mySphereID], gran_params),
        gran_params);

    sphere_data_struct mydata = sphere_info[mySphereID];

    // The value is the sphere id, to be sorted with owner as the key
    mydata.pos_X = d_sphere_pos_X[mySphereID];
    mydata.pos_Y = d_sphere_pos_Y[mySphereID];
    mydata.pos_Z = d_sphere_pos_Z[mySphereID];
    mydata.pos_X_dt = d_sphere_pos_X_dt[mySphereID];
    mydata.pos_Y_dt = d_sphere_pos_Y_dt[mySphereID];
    mydata.pos_Z_dt = d_sphere_pos_Z_dt[mySphereID];
    // replace with the new data
    sphere_info[mySphereID] = mydata;
}
// unpack sorted data to global memory, very coalesced
template <unsigned int CUB_THREADS>
__global__ void owner_unpack(int* d_sphere_pos_X,
                             int* d_sphere_pos_Y,
                             int* d_sphere_pos_Z,
                             float* d_sphere_pos_X_dt,
                             float* d_sphere_pos_Y_dt,
                             float* d_sphere_pos_Z_dt,
                             unsigned int nSpheres,
                             sphere_data_struct* sphere_info,
                             ParamsPtr gran_params) {
    // Figure out what sphereID this thread will handle. We work with a 1D block structure and a 1D grid structure
    unsigned int mySphereID = threadIdx.x + blockIdx.x * blockDim.x;
    // Only do this for valid spheres
    if (mySphereID >= nSpheres) {
        return;
    }
    sphere_data_struct mydata = sphere_info[mySphereID];

    // The value is the sphere id, to be sorted with owner as the key
    d_sphere_pos_X[mySphereID] = mydata.pos_X;
    d_sphere_pos_Y[mySphereID] = mydata.pos_Y;
    d_sphere_pos_Z[mySphereID] = mydata.pos_Z;
    d_sphere_pos_X_dt[mySphereID] = mydata.pos_X_dt;
    d_sphere_pos_Y_dt[mySphereID] = mydata.pos_Y_dt;
    d_sphere_pos_Z_dt[mySphereID] = mydata.pos_Z_dt;
}

// Sorts data by owner SD, makes nicer memory accesses
// Uses a boatload of memory
__host__ void ChSystemGranularMonodisperse_SMC::defragment_data() {
    VERBOSE_PRINTF("Starting defrag run!\n");
    unsigned int nBlocks = (nDEs + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    // Set of pointers for each buffer
    unsigned int* d_owners;
    sphere_data_struct* d_sphere_data;
    // second buffer for nice sort
    unsigned int* d_owners_2;
    sphere_data_struct* d_sphere_data_2;
    // Allocate some nice memory
    gpuErrchk(hipMalloc(&d_owners, nDEs * sizeof(unsigned int)));
    gpuErrchk(hipMalloc(&d_sphere_data, nDEs * sizeof(sphere_data_struct)));
    gpuErrchk(hipMalloc(&d_owners_2, nDEs * sizeof(unsigned int)));
    gpuErrchk(hipMalloc(&d_sphere_data_2, nDEs * sizeof(sphere_data_struct)));
    owner_prepack<CUDA_THREADS_PER_BLOCK><<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(
        pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data(), nDEs, d_owners,
        d_sphere_data, gran_params);
    gpuErrchk(hipDeviceSynchronize());

    // Create a set of DoubleBuffers to wrap pairs of device pointers
    hipcub::DoubleBuffer<unsigned int> d_keys(d_owners, d_owners_2);
    hipcub::DoubleBuffer<sphere_data_struct> d_values(d_sphere_data, d_sphere_data_2);

    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    // pass null, cub tells us what it needs
    hipcub::DeviceRadixSort::SortPairs(NULL, temp_storage_bytes, d_keys, d_values, nDEs);
    gpuErrchk(hipDeviceSynchronize());

    // Allocate temporary storage
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    gpuErrchk(hipDeviceSynchronize());

    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, nDEs);
    gpuErrchk(hipDeviceSynchronize());

    owner_unpack<CUDA_THREADS_PER_BLOCK>
        <<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt.data(),
                                              pos_Y_dt.data(), pos_Z_dt.data(), nDEs, d_values.Current(), gran_params);
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_owners);
    hipFree(d_owners_2);
    hipFree(d_sphere_data);
    hipFree(d_sphere_data_2);
    hipFree(d_temp_storage);
    VERBOSE_PRINTF("defrag finished!\n");
}

__global__ void generate_absv(const unsigned int nDEs,
                              const float* velX,
                              const float* velY,
                              const float* velZ,
                              float* d_absv) {
    unsigned int my_sphere = blockIdx.x * blockDim.x + threadIdx.x;
    if (my_sphere < nDEs) {
        float v[3] = {velX[my_sphere], velY[my_sphere], velZ[my_sphere]};
        d_absv[my_sphere] = v[0] * v[0] + v[1] * v[1] + v[2] * v[2];
    }
}

__host__ float ChSystemGranular::get_max_vel() {
    float* d_absv;
    float* d_max_vel;
    float h_max_vel;
    gpuErrchk(hipMalloc(&d_absv, nDEs * sizeof(float)));
    gpuErrchk(hipMalloc(&d_max_vel, sizeof(float)));

    generate_absv<<<(nDEs + 255) / 256, 256>>>(nDEs, pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data(), d_absv);

    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_absv, d_max_vel, nDEs);
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_absv, d_max_vel, nDEs);
    gpuErrchk(hipMemcpy(&h_max_vel, d_max_vel, sizeof(float), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_absv));
    gpuErrchk(hipFree(d_max_vel));

    return h_max_vel;
}

__host__ void ChSystemGranularMonodisperse_SMC::initialize() {
    switch_to_SimUnits();
    generate_DEs();

    // Set aside memory for holding data structures worked with. Get some initializations going
    setup_simulation();
    copy_const_data_to_device();
    copyBD_Frame_to_device();
    gpuErrchk(hipDeviceSynchronize());

    determine_new_stepSize_SU();

    // Seed arrays that are populated by the kernel call
    resetBroadphaseInformation();

    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nDEs + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
    printf("doing priming!\n");
    printf("max possible composite offset is %zu\n", (size_t)nSDs * MAX_COUNT_OF_DEs_PER_SD);

    auto sphere_data = packSphereDataPointers();

    primingOperationsRectangularBox<CUDA_THREADS_PER_BLOCK>
        <<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data, nDEs, gran_params);
    gpuErrchk(hipDeviceSynchronize());
    printf("priming finished!\n");

    printf("z grav term with timestep %f is %f\n", stepSize_SU, stepSize_SU * stepSize_SU * gravity_Z_SU);
    printf("running at approximate timestep %f\n", stepSize_SU * gran_params->TIME_UNIT);
}

__host__ double ChSystemGranularMonodisperse_SMC::advance_simulation(float duration) {
    auto sphere_data = packSphereDataPointers();

    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nDEs + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    // Settling simulation loop.
    float duration_SU = std::ceil(duration / gran_params->TIME_UNIT);
    unsigned int nsteps = duration_SU / stepSize_SU;

    VERBOSE_PRINTF("advancing by %f at timestep %f, %u timesteps at approx user timestep %f\n", duration_SU,
                   stepSize_SU, nsteps, duration / nsteps);
    float time_elapsed_SU = 0;  // time elapsed in this advance call
    // Run the simulation, there are aggressive synchronizations because we want to have no race conditions
    for (; time_elapsed_SU < stepSize_SU * nsteps; time_elapsed_SU += stepSize_SU) {
        determine_new_stepSize_SU();  // doesn't always change the timestep

        gran_params->alpha_h_bar = stepSize_SU;
        // Update the position and velocity of the BD, if relevant
        if (!BD_is_fixed) {
            updateBDPosition(stepSize_SU);  // TODO current time
        }
        resetSphereForces();

        VERBOSE_PRINTF("Starting computeSphereForces!\n");

        // Compute sphere-sphere forces
        computeSphereForces<MAX_COUNT_OF_DEs_PER_SD><<<nSDs, MAX_COUNT_OF_DEs_PER_SD>>>(
            sphere_data, gran_params, BC_type_list.data(), BC_params_list.data(), BC_params_list.size());

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        VERBOSE_PRINTF("Resetting broadphase info!\n");

        resetBroadphaseInformation();

        VERBOSE_PRINTF("Starting updatePositions!\n");
        updatePositions<CUDA_THREADS_PER_BLOCK>
            <<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(stepSize_SU, sphere_data, nDEs, gran_params);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        elapsedSimTime += stepSize_SU * gran_params->TIME_UNIT;  // Advance current time
    }

    return time_elapsed_SU * gran_params->TIME_UNIT;  // return elapsed UU time
}
}  // namespace granular
}  // namespace chrono
