#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <climits>
#include "../../chrono_thirdparty/hipcub/hipcub.hpp"
#include "../ChGranularDefines.h"
#include "assert.h"

#define BLAH_F 0.f
#define BLAH_I 0
#define NULL_SD UINT_MAX-1 
#define NULL_SphID UINT_MAX-1 

__device__ dim3 whereSphCenterIs(const float* const sphereXYZ, const float3& xyzOriginBox, const float4& eulerParamBox, const ushort3& SDdims, const dim3& RectangularBoxDims)
{
    return BLAH_I;
}

__device__ size_t figureOutTouchedSDs(const dim3& , unsigned int* touchedSDs);

/**
* This kernel call prepares information that will be used in a subsequent kernel that performs the actual time stepping.
* 
* Assumptions:
*   - Granular material is made up of spheres. 
*   - For now, all spheres are of constant radius. The radius of the sphere is 1.f
*   - The function below assumes the spheres are in a box
*   - The box has dimensions L x W x H. 
*   - The reference frame associated with the box:
*       - The x-axis is along the length L of the box
*       - The y-axis is along the width W of the box
*       - The z-axis is along the height H of the box
*   - A sphere cannot touch more than eight SDs
*   - The total number of SDs touched by the sphres worked upon in this CUDA block is less than USHRT_MAX. This is 
*       reasonable given that we are not going to have more than 1024 spheres worked upon in a CUDA block
*
* Basic idea: use domain decomposition on the rectangular box and have one subdomain be processed by one block.
* The subdomains are axis-aligned relative to the reference frame associated with the *box*. The origin of the box is at the center of the box.
* The orientation of the box is defined relative to a world inertial reference frame.
*
* Nomenclature:
*   - SD: subdomain.
*   - BD: the big-domain, which is the union of all SDs
*
* Notes:
*   - The SD with ID=0 is the catch-all SD. This is the SD in which a sphere ends up if its not inside the rectangular box. Usually, there is no sphere in this SD
*
*
*/
template<unsigned short int BLOCK_THREADS, unsigned short int SHMEM_UINT, unsigned short int SHMEM_FL>
__global__ void primingOperationsRectangularBox(
    float3 xyzOriginBox,                      //!< Set of three floats that give the location of the rectangular box in the Global Reference Frame
    float4 eulerParamBox,                     //!< Set of four floats that provide the orientation of the rectangular box in the Global Reference Frame
    ushort3 SD_dims,                          //!< Set of three ints that provide the dimension (in multiple of Sphere radia) of a SD
    dim3 RectangularBox_dims,                 //!< The dimension of the rectangular box. The 3D box is expressed in multpiples of SD, in the X, Y, and Z directions, respectively
    float* pRawDataArray,                     //!< Pointer to array containing data related to the spheres in the box
    unsigned int* SD_countsOfSheresTouching,  //!< The array that for each SD indicates how many spheres touch this SD
    unsigned int* spheres_in_SD_composite,    //!< Big array that works in conjunction with SD_countsOfSheresTouching. "spheres_in_SD_composite" says which SD contains what spheres
    size_t nSpheres                           //!< Number of spheres in the box
)
{
    /// Set aside some shared memory (need to look how this gets word aligned)
    __shared__ unsigned int shMem_UINT[SHMEM_UINT];
    __shared__ float shMem_float[SHMEM_FL];


    // We work with a 1D block structure and a 3D grid structure
    unsigned int mySphereID = threadIdx.x + blockIdx.x * blockDim.x * blockDim.y * blockDim.z;

    unsigned short int N_SDsTouched = 0;           /// stores the number of SDs touched by this sphere
    float sphereXYZ[3];                            /// the coordinates of the sphere
    unsigned int touchedSDs[8] = { NULL_SD, NULL_SD, NULL_SD, NULL_SD, NULL_SD, NULL_SD, NULL_SD, NULL_SD };  /// The equivalent of a non-touchable SD
    unsigned int sphereID  [8] = { NULL_SphID, NULL_SphID, NULL_SphID, NULL_SphID, NULL_SphID, NULL_SphID, NULL_SphID, NULL_SphID };  /// The equivalent of a non-existent sphere
    if (mySphereID < nSpheres) {
        // Bring the "center of sphere" information via CUB
        typedef hipcub::BlockLoad<float, BLOCK_THREADS, 3, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> BlockLoad;
        // Allocate shared memory for BlockLoad
        __shared__ typename BlockLoad::TempStorage temp_storage;
        // Load a segment of consecutive items that are blocked across threads
        BlockLoad(temp_storage).Load(pRawDataArray + 3*mySphereID, sphereXYZ);

        // Find out which SDs are touched by this sphere
        // NOTE: A sphere might also touch the "catchAll_SD"
        // "catchAll_SD": subdomain that encompasses all the universe except the RectangularBox of interest
        dim3 whichSD_SphCenterIsIn = whereSphCenterIs(sphereXYZ, xyzOriginBox, eulerParamBox, SD_dims, RectangularBox_dims);
        N_SDsTouched = figureOutTouchedSDs(whichSD_SphCenterIsIn, touchedSDs); ///NOT DONE YET

        // Load the proper value, to be used later for key-value sort
        sphereID[0] = mySphereID;
        sphereID[1] = mySphereID;
        sphereID[2] = mySphereID;
        sphereID[3] = mySphereID;
        sphereID[4] = mySphereID;
        sphereID[5] = mySphereID;
        sphereID[6] = mySphereID;
        sphereID[7] = mySphereID;
    }

    __syncthreads();

    // Do a collective SIMT operation: "reduce_op" to figure out how much memory needs to be set aside  
    __shared__ unsigned short int totalNumberOfSphere_SD_touches;
    typedef hipcub::BlockReduce<unsigned short int, BLOCK_THREADS, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduceT;
    __shared__ typename BlockReduceT::TempStorage temp_storage;
    unsigned short int dummy = BlockReduceT(temp_storage).Sum(N_SDsTouched);
    if (threadIdx.x == 0) {
        totalNumberOfSphere_SD_touches = dummy;
    }

    // Do a sort by key, sorting in increasing order; note that the entries that store the 
    // untouchable SD; i.e., NULL_SD, will migrate to the end of the array
    // The key: the domain touched by this Sphere
    // The value: the sphere ID
    typedef hipcub::BlockRadixSort<unsigned int, BLOCK_THREADS, 8, unsigned int> BlockRadixSort;
    __shared__ typename BlockRadixSort::TempStorage temp_storage_sort;
    BlockRadixSort(temp_storage_sort).Sort(touchedSDs, sphereID);

    // Stitch together in ShMem the unsorted array containing the SDs touched by the sphere handled in this block
    // Specialize BlockStore for a 1D block using BLOCK_THREADS threads each and owning 8 unsigned integer items each
    typedef hipcub::BlockStore<unsigned int, BLOCK_THREADS, 8, hipcub::BLOCK_STORE_VECTORIZE > BlockStore; // make sure address is quadword-aligned
    // Allocate shared memory for BlockStore
    __shared__ typename BlockStore::TempStorage temp_storage_stitching;
    BlockStore(temp_storage_stitching).Store(shMem_UINT, touchedSDs);

    // Figure out what each thread needs to check in terms of Heaviside-step in the array of SDs 
    // touched; the BLOCK_THREADS is promoted to unsigned int here
    unsigned int dummyUINT = (totalNumberOfSphere_SD_touches + BLOCK_THREADS - 1) / BLOCK_THREADS;

    unsigned int startPoint = threadIdx.x * dummyUINT;
    unsigned int endPoint   = startPoint + dummyUINT;

    // From spheres with IDs between startPoint to endPoint, figure out in which SD each sphere goes and 
    // do prep work to place it in there. No need to synchronize the block threads since although we
    // write to shared memory, we only work within the startPoint to endPoint bounds. There is some 
    // atomicAdd overhead, but that's going to be cached in L2 --> something to look into in the future
    unsigned int ref_SDid = shMem_UINT[startPoint];
    unsigned int n_repetitions = 0;
    for (unsigned int i = startPoint; i < endPoint; i++) {
        if (ref_SDid == shMem_UINT[i]) {
            // This is guaranteed to be hit for i=startPoint
            n_repetitions++;
        }
        else {
            dummyUINT = atomicAdd(SD_countsOfSheresTouching + ref_SDid, n_repetitions);
            ref_SDid = i - n_repetitions; // using ref_SDid as a dummy; soon to be set to something meaningful
            do {
                // Compute the offset in the SD data array where this sphere should deposit its information.
                // The "this sphere" is the sphere with ID stored in sphereID[i]. Note that we're 
                // overwritting what used to be in shMem_UINT; i.e., which SD this sphere touches, since
                // this information is not needed anymore
                shMem_UINT[ref_SDid++] = dummyUINT++; // exptected to give a coalesced mem access later on
                n_repetitions--;
            } while (n_repetitions > 0);
            ref_SDid = shMem_UINT[i]; // we have a new reference at this point; 
            n_repetitions = 1;        // set to 1 since each SD is hit at least once
        }
    }

    __syncthreads(); // Wating here on all threads to finish before writing the data to global memory.

    if (threadIdx.x < totalNumberOfSphere_SD_touches)
        spheres_in_SD_composite[shMem_UINT[threadIdx.x]] = sphereID[threadIdx.x];
    
    return;
}