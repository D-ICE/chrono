#include "hip/hip_runtime.h"
﻿// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2018 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Dan Negrut, Conlain Kelly
// =============================================================================

#include <hip/hip_runtime.h>
#include <cstdio>
#include <fstream>
#include "../../chrono_thirdparty/hipcub/hipcub.hpp"
#include "../ChGranularDefines.h"
#include "../chrono_granular/physics/ChGranular.h"
#include "chrono_granular/physics/ChGranularDefines.cuh"
#include "chrono_granular/utils/ChGranularUtilities_CUDA.cuh"

__constant__ unsigned int d_monoDisperseSphRadius_SU;  //!< Radius of the sphere, expressed in SU

__constant__ unsigned int d_SD_Ldim_SU;    //!< Ad-ed L-dimension of the SD box
__constant__ unsigned int d_SD_Ddim_SU;    //!< Ad-ed D-dimension of the SD box
__constant__ unsigned int d_SD_Hdim_SU;    //!< Ad-ed H-dimension of the SD box
__constant__ unsigned int psi_T_dFactor;   //!< factor used in establishing the software-time-unit
__constant__ unsigned int psi_h_dFactor;   //!< factor used in establishing the software-time-unit
__constant__ unsigned int psi_L_dFactor;   //!< factor used in establishing the software-time-unit
__constant__ unsigned int d_box_L_SU;      //!< Ad-ed L-dimension of the BD box in multiples of subdomains
__constant__ unsigned int d_box_D_SU;      //!< Ad-ed D-dimension of the BD box in multiples of subdomains
__constant__ unsigned int d_box_H_SU;      //!< Ad-ed H-dimension of the BD box in multiples of subdomains
__constant__ float gravAcc_X_d_factor_SU;  //!< Device counterpart of the constant gravAcc_X_factor_SU
__constant__ float gravAcc_Y_d_factor_SU;  //!< Device counterpart of the constant gravAcc_Y_factor_SU
__constant__ float gravAcc_Z_d_factor_SU;  //!< Device counterpart of the constant gravAcc_Z_factor_SU

/// Takes in a sphere's position and inserts into the given int array[8] which subdomains, if any, are touched
/// The array is indexed with the ones bit equal to +/- x, twos bit equal to +/- y, and the fours bit equal to +/- z
/// A bit set to 0 means the lower index, whereas 1 means the higher index (lower + 1)
/// The kernel computes global x, y, and z indices for the bottom-left subdomain and then uses those to figure out which
/// subdomains described in the corresponding 8-SD cube are touched by the sphere. The kernel then converts these
/// indices to indices into the global SD list via the (currently local) conv[3] data structure
/// Should be mostly bug-free, especially away from boundaries
__device__ void figureOutTouchedSD(int sphCenter_X, int sphCenter_Y, int sphCenter_Z, unsigned int SDs[8]) {
    // I added these to fix a bug, we can inline them if/when needed but they ARE necessary
    // We need to offset so that the bottom-left corner is at the origin
    int sphCenter_X_modified = (d_box_L_SU * d_SD_Ldim_SU) / 2 + sphCenter_X;
    int sphCenter_Y_modified = (d_box_D_SU * d_SD_Ddim_SU) / 2 + sphCenter_Y;
    int sphCenter_Z_modified = (d_box_H_SU * d_SD_Hdim_SU) / 2 + sphCenter_Z;
    int n[3];
    // TODO this doesn't handle if the ball is slightly penetrating the boundary, could result in negative values or end
    // GIDs beyond bounds. We might want to do a check to see if it's outside and set 'valid' accordingly
    // NOTE: This is integer arithmetic to compute the floor. We want to get the first SD below the sphere
    // nx = (xCenter - radius) / wx .
    n[0] = (sphCenter_X_modified - d_monoDisperseSphRadius_SU) / d_SD_Ldim_SU;
    // Same for D and H
    n[1] = (sphCenter_Y_modified - d_monoDisperseSphRadius_SU) / d_SD_Ddim_SU;
    n[2] = (sphCenter_Z_modified - d_monoDisperseSphRadius_SU) / d_SD_Hdim_SU;
    // Find distance from next box in relevant dir to center, we may be straddling the two
    int d[3];                                                 // Store penetrations
    d[0] = (n[0] + 1) * d_SD_Ldim_SU - sphCenter_X_modified;  // dx = (nx + 1)* wx - x
    d[1] = (n[1] + 1) * d_SD_Ddim_SU - sphCenter_Y_modified;
    d[2] = (n[2] + 1) * d_SD_Hdim_SU - sphCenter_Z_modified;

    // Calculate global indices from locals
    // ones bit is x, twos bit is y, threes bit is z
    // do some cute bit shifting and snag bit at correct place
    // For each index in SDs
    for (int i = 0; i < 8; i++) {
        SDs[i] = 0;                // Init to 0
        unsigned int valid = 0x1;  // Assume this SD is touched at start

        // s adds an offset to directional index for SDs
        // High/low in x-dir
        // unsigned int s = i & 0x1; // Inlined now
        // Scale to global index and add to total
        SDs[i] += (n[0] + (i & 0x1)) * d_box_D_SU * d_box_H_SU;
        // s == own[e] evals true if the current SD is owner
        // If both touch it or we own it, the result is valid
        valid &= (abs(d[0]) < d_monoDisperseSphRadius_SU) || ((i & 0x1) == (d[0] < 0));

        // High/low in y-dir
        // s = i & 0x2; // Inlined now
        // Scale to global index and add to total
        SDs[i] += (n[1] + ((i >> 1) & 0x1)) * d_box_H_SU;
        // If both touch it or we own it, the result is valid
        valid &= (abs(d[1]) < d_monoDisperseSphRadius_SU) || (((i >> 1) & 0x1) == (d[1] < 0));

        // High/low in z-dir
        // s = i & 0x4; // Inlined now
        // Scale to global index and add to total
        SDs[i] += (n[2] + ((i >> 2) & 0x1));
        // If both touch it or we own it, the result is valid
        valid &= (abs(d[2]) < d_monoDisperseSphRadius_SU) || (((i >> 2) & 0x1) == (d[2] < 0));

        // This ternary is hopefully better than a conditional
        // If valid is false, then the SD is actually NULL_GRANULAR_ID
        SDs[i] = (valid ? SDs[i] : NULL_GRANULAR_ID);
    }
}
/**
 * This kernel call prepares information that will be used in a subsequent kernel that performs the actual time
 * stepping.
 *
 * Template arguments:
 *   - CUB_THREADS: the number of threads used in this kernel, comes into play when invoking CUB block collectives
 *
 * Assumptions:
 *   - Granular material is made up of monodisperse spheres.
 *   - The function below assumes the spheres are in a box
 *   - The box has dimensions L x D x H.
 *   - The reference frame associated with the box:
 *       - The x-axis is along the length L of the box
 *       - The y-axis is along the width D of the box
 *       - The z-axis is along the height H of the box
 *   - A sphere cannot touch more than eight SDs
 *
 * Basic idea: use domain decomposition on the rectangular box and figure out how many SDs each sphere touches.
 * The subdomains are axis-aligned relative to the reference frame associated with the *box*. The origin of the box is
 * at the center of the box. The orientation of the box is defined relative to a world inertial reference frame.
 *
 * Nomenclature:
 *   - SD: subdomain.
 *   - BD: the big-domain, which is the union of all SDs
 *   - NULL_GRANULAR_ID: the equivalent of a non-sphere SD ID, or a non-sphere ID
 *
 * Notes:
 *   - The SD with ID=0 is the catch-all SD. This is the SD in which a sphere ends up if its not inside the rectangular
 * box. Usually, there is no sphere in this SD (THIS IS NOT IMPLEMENTED AS SUCH FOR NOW)
 *
 */
template <
    unsigned int
        CUB_THREADS>  //!< Number of CUB threads engaged in block-collective CUB operations. Should be a multiple of 32
__global__ void
primingOperationsRectangularBox(
    int* pRawDataX,                            //!< Pointer to array containing data related to the spheres in the box
    int* pRawDataY,                            //!< Pointer to array containing data related to the spheres in the box
    int* pRawDataZ,                            //!< Pointer to array containing data related to the spheres in the box
    unsigned int* SD_countsOfSpheresTouching,  //!< The array that for each SD indicates how many spheres touch this SD
    unsigned int* spheres_in_SD_composite,     //!< Big array that works in conjunction with SD_countsOfSpheresTouching.
                                               //!< "spheres_in_SD_composite" says which SD contains what spheres
    unsigned int nSpheres                      //!< Number of spheres in the box
) {
    int xSphCenter;
    int ySphCenter;
    int zSphCenter;

    /// Set aside shared memory
    volatile __shared__ unsigned int offsetInComposite_SphInSD_Array[CUB_THREADS * 8];
    volatile __shared__ bool shMem_head_flags[CUB_THREADS * 8];

    typedef hipcub::BlockRadixSort<unsigned int, CUB_THREADS, 8, unsigned int> BlockRadixSortOP;
    __shared__ typename BlockRadixSortOP::TempStorage temp_storage_sort;

    typedef hipcub::BlockDiscontinuity<unsigned int, CUB_THREADS> Block_Discontinuity;
    __shared__ typename Block_Discontinuity::TempStorage temp_storage_disc;

    // Figure out what sphereID this thread will handle. We work with a 1D block structure and a 1D grid structure
    unsigned int mySphereID = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int sphIDs[8] = {mySphereID, mySphereID, mySphereID, mySphereID,
                              mySphereID, mySphereID, mySphereID, mySphereID};

    // This uses a lot of registers but is needed
    unsigned int SDsTouched[8] = {NULL_GRANULAR_ID, NULL_GRANULAR_ID, NULL_GRANULAR_ID, NULL_GRANULAR_ID,
                                  NULL_GRANULAR_ID, NULL_GRANULAR_ID, NULL_GRANULAR_ID, NULL_GRANULAR_ID};
    if (mySphereID < nSpheres) {
        // Coalesced mem access
        xSphCenter = pRawDataX[mySphereID];
        ySphCenter = pRawDataY[mySphereID];
        zSphCenter = pRawDataZ[mySphereID];

        figureOutTouchedSD(xSphCenter, ySphCenter, zSphCenter, SDsTouched);
    }

    __syncthreads();

    // Sort by the ID of the SD touched
    BlockRadixSortOP(temp_storage_sort).Sort(SDsTouched, sphIDs);
    __syncthreads();

    // Do a winningStreak search on whole block, might not have high utilization here
    bool head_flags[8];
    Block_Discontinuity(temp_storage_disc).FlagHeads(head_flags, SDsTouched, hipcub::Inequality());
    __syncthreads();

    // Write back to shared memory; eight-way bank conflicts here - to revisit later
    for (unsigned int i = 0; i < 8; i++) {
        shMem_head_flags[8 * threadIdx.x + i] = head_flags[i];
    }

    // Seed offsetInComposite_SphInSD_Array with "no valid ID" so that we know later on what is legit;
    // No shmem bank coflicts here, good access...
    for (unsigned int i = 0; i < 8; i++) {
        offsetInComposite_SphInSD_Array[i * CUB_THREADS + threadIdx.x] = NULL_GRANULAR_ID;
    }

    __syncthreads();

    // Count how many times an SD shows up in conjunction with the collection of CUB_THREADS spheres. There
    // will be some thread divergence here.
    // Loop through each potential SD, after sorting, and see if it is the start of a head
    for (unsigned int i = 0; i < 8; i++) {
        // SD currently touched, could easily be inlined
        unsigned int touchedSD = SDsTouched[i];
        if (touchedSD != NULL_GRANULAR_ID && head_flags[i]) {
            // current index into shared datastructure of length 8*CUB_THREADS, could easily be inlined
            unsigned int idInShared = 8 * threadIdx.x + i;
            unsigned int winningStreak = 0;
            // This is the beginning of a sequence of SDs with a new ID
            do {
                winningStreak++;
                // Go until we run out of threads on the warp or until we find a new head
            } while (idInShared + winningStreak < 8 * CUB_THREADS && !(shMem_head_flags[idInShared + winningStreak]));

            // if (touchedSD >= d_box_L_SU * d_box_D_SU * d_box_H_SU) {
            //     printf("invalid SD index %u on thread %u\n", mySphereID, touchedSD);
            // }

            // Store start of new entries
            unsigned int offset = atomicAdd(SD_countsOfSpheresTouching + touchedSD, winningStreak);

            // The value offset now gives a *relative* offset in the composite array; i.e., spheres_in_SD_composite.
            // Get the absolute offset
            offset += touchedSD * MAX_COUNT_OF_DEs_PER_SD;

            // Produce the offsets for this streak of spheres with identical SD ids
            for (unsigned int i = 0; i < winningStreak; i++)
                offsetInComposite_SphInSD_Array[idInShared + i] = offset++;
        }
    }

    __syncthreads();  // needed since we write to shared memory above; i.e., offsetInComposite_SphInSD_Array

    // Write out the data now; reister with spheres_in_SD_composite each sphere that touches a certain ID
    for (unsigned int i = 0; i < 8; i++) {
        unsigned int offset = offsetInComposite_SphInSD_Array[8 * threadIdx.x + i];
        if (offset != NULL_GRANULAR_ID)
            spheres_in_SD_composite[offset] = sphIDs[i];
    }
}
/// Count the number of contacts for each body
/// TODO this does more work than strictly necessary, but is much cleaner
/// we could probably get up 2x speedup on this function call by some cute mapping, but that would require a global map
template <unsigned int MAX_NSPHERES_PER_SD>
__device__ unsigned int dryRunContactCount(unsigned tIdx,
                                           const int sph_X[MAX_NSPHERES_PER_SD],
                                           const int sph_Y[MAX_NSPHERES_PER_SD],
                                           const int sph_Z[MAX_NSPHERES_PER_SD]) {
    unsigned int ncontacts = 0;  // We return this value
    // This function call returns the number of contacts
    unsigned int sphere1 = tIdx;
    for (unsigned int sphere2 = 0; sphere2 < MAX_NSPHERES_PER_SD; sphere2++) {
        // Check both sphere for legal values
        // If either sphere is invalid or the sphere to check has lower index, skip this check
        // Store boolean as uint because why not
        unsigned int invalid =
            (sph_X[sphere1] == ILL_GRANULAR_VAL) || (sph_X[sphere2] == ILL_GRANULAR_VAL) || (sphere1 >= sphere2);
        unsigned int dx = (sph_X[sphere1] - sph_X[sphere2]);
        unsigned int dy = (sph_Y[sphere1] - sph_Y[sphere2]);
        unsigned int dz = (sph_Z[sphere1] - sph_Z[sphere2]);
        unsigned int d2 = dx * dx + dy * dy + dz * dz;
        // True if bodies are in contact, true->1 in c++
        unsigned int contact = (d2 < d_monoDisperseSphRadius_SU * d_monoDisperseSphRadius_SU);
        // Store so we can return it later
        ncontacts += contact && !invalid;
    }

    return ncontacts;
}

// Do the same work as dryrun, but load the sphere IDs this time
template <unsigned int MAX_NSPHERES_PER_SD>
__device__ void populateContactEventInformation_dataStructures(
    unsigned tIdx,
    const int sph_X[MAX_NSPHERES_PER_SD],
    const int sph_Y[MAX_NSPHERES_PER_SD],
    const int sph_Z[MAX_NSPHERES_PER_SD],
    unsigned int thisThrdOffset,
    unsigned int thisThrdCollisionCount,
    volatile unsigned char IDfrstDE_inCntctEvent[MAX_NSPHERES_PER_SD * AVERAGE_COUNT_CONTACTS_PER_DE],
    volatile unsigned char IDscndDE_inCntctEvent[MAX_NSPHERES_PER_SD * AVERAGE_COUNT_CONTACTS_PER_DE]) {
    unsigned int count = 0;  // use this as index into shared memory
    // This function call returns the number of contacts
    unsigned int sphere1 = tIdx;
    // this code shouldn't diverge too much, at worst it will run the 'else' of the if case MAX_NSPHERES_PER_SD time
    for (unsigned int sphere2 = 0; sphere2 < MAX_NSPHERES_PER_SD; sphere2++) {
        // Check both sphere for legal values
        // If either sphere is invalid or the sphere to check has lower index, skip this check
        // Store boolean as uint because it sneaks past conditional checks that way
        unsigned int invalid =
            (sph_X[sphere1] == ILL_GRANULAR_VAL) || (sph_X[sphere2] == ILL_GRANULAR_VAL) || (sphere1 >= sphere2);
        unsigned int dx = (sph_X[sphere1] - sph_X[sphere2]);
        unsigned int dy = (sph_Y[sphere1] - sph_Y[sphere2]);
        unsigned int dz = (sph_Z[sphere1] - sph_Z[sphere2]);
        unsigned int d2 = dx * dx + dy * dy + dz * dz;
        // True if bodies are in contact, true->1 in c++
        unsigned int contact = (d2 < d_monoDisperseSphRadius_SU * d_monoDisperseSphRadius_SU);
        // This is warp divergence but it shouldn't be the _worst_
        if (contact && !invalid) {
            // Write back to shared memory
            IDfrstDE_inCntctEvent[tIdx + count] = sphere1;
            IDscndDE_inCntctEvent[tIdx + count] = sphere2;
            // Increment counter
            count++;
            if (count > thisThrdCollisionCount) {
                printf("BIG BIG ERROR! too many collisions detected!\n");
            }
        }
    }
}

/**
This device function computes the forces induces by the walls on the box on a sphere
Input:
  - sphXpos: X location, measured in the box reference system, of the sphere
  - sphYpos: Y location, measured in the box reference system, of the sphere
  - sphZpos: Z location, measured in the box reference system, of the sphere

Output:
  - Xforce: the X component of the force, as represented in the box reference system
  - Yforce: the Y component of the force, as represented in the box reference system
  - Zforce: the Z component of the force, as represented in the box reference system
*/
__device__ void boxWallsEffects(int sphXpos, int sphYpos, int sphZpos, float& Xforce, float& Yforce, float& Zforce) {
    Xforce = 0.f;
    Yforce = 0.f;
    Zforce = 0.f;
}

/**
This device function figures out how many contact events the thread "thrdIndx" needs to take care of.
Input:
    - tIdx: the thread for which we identify the work order
    - blockLvlCollisionEventsCount: the total number of contact events the entire block needs to deal with
Output:
    - myColsnCount: the number of contact events that thread thrdIndx will have to deal with
    - my_offset: offset in the collision data structure where this thread starts
*/
template <unsigned int MAX_NSPHERES_PER_SD>
__device__ void figureOutWorkOrder(unsigned int tIdx,
                                   unsigned int blockLvlCollisionEventsCount,
                                   unsigned int* myColsnCount,
                                   unsigned int* my_offset) {
    const unsigned int num_threads = blockDim.x;  // Get this info
    // We want to calculate the number of collisions per thread, but not undershoot
    unsigned int collisions_per_thread = (blockLvlCollisionEventsCount + num_threads - 1) / num_threads;
    // the number of extra collisions we picked up
    unsigned int rem = num_threads * collisions_per_thread - blockLvlCollisionEventsCount;
    // Offset into total collisions, this should be the same for everyone
    *my_offset = tIdx * collisions_per_thread;
    // Last rem threads should do less work
    // Should be little warp divergence since we're only doing one check and this is a tiny function anyways
    if (tIdx >= num_threads - rem) {
        // Don't do the extra work on this thread
        collisions_per_thread -= 1;
    }
    // Store this thread's work to do
    *myColsnCount = collisions_per_thread;
}

/**
This device function computes the normal force between two spheres that are in contact.
Input:
- delX: the difference in the x direction between the the two spheres; i.e., x_A - x_B
- delY: the difference in the y direction between the the two spheres; i.e., y_A - y_B
- delZ: the difference in the z direction between the the two spheres; i.e., z_A - z_B
Output:
- sphA_Xforce: the force that sphere A "feels" in the x direction
- sphA_Yforce: the force that sphere A "feels" in the y direction
- sphA_Zforce: the force that sphere A "feels" in the z direction
*/
__device__ void computeNormalForce(const int& delX,
                                   const int& delY,
                                   const int& delZ,
                                   int& sphA_Xforce,
                                   int& sphA_Yforce,
                                   int& sphA_Zforce) {
    sphA_Xforce = ILL_GRANULAR_VAL;
    sphA_Yforce = ILL_GRANULAR_VAL;
    sphA_Zforce = ILL_GRANULAR_VAL;
}

/**
This kernel call figures out forces on a sphere and carries out numerical integration to get the velocities of a sphere.

Template arguments:
  - MAX_NSPHERES_PER_SD: the number of threads used in this kernel, comes into play when invoking CUB block collectives.
                         NOTE: It is assumed that MAX_NSPHERES_PER_SD<256 (we are using in this kernel unsigned char to
store IDs)

Assumptions:
  - Granular material is made up of monodisperse spheres.
  - The function below assumes the spheres are in a box
  - The box has dimensions L x D x H.
  - The reference frame associated with the box:
      - The x-axis is along the length L of the box
      - The y-axis is along the width D of the box
      - The z-axis is along the height H of the box
  - A sphere cannot touch more than eight SDs

Basic idea: use domain decomposition on the rectangular box and figure out how many SDs each sphere touches.
The subdomains are axis-aligned relative to the reference frame associated with the *box*. The origin of the box is
at the center of the box. The orientation of the box is defined relative to a world inertial reference frame.

Nomenclature:
  - SD: subdomain.
  - NULL_GRANULAR_ID: the equivalent of a non-sphere SD ID, or a non-sphere ID

Notes:
  - The SD with ID=0 is the catch-all SD. This is the SD in which a sphere ends up if its not inside the rectangular
box. Usually, there is no sphere in this SD (THIS IS NOT IMPLEMENTED AS SUCH FOR NOW)
*/
template <unsigned int MAX_NSPHERES_PER_SD>  //!< Number of CUB threads engaged in block-collective CUB operations.
                                             //!< Should be a multiple of 32
__global__ void updateVelocities(unsigned int alpha_h_bar,  //!< Value that controls actual step size.
                                 int* pRawDataX,            //!< Pointer to array containing data related to the
                                                            //!< spheres in the box
                                 int* pRawDataY,            //!< Pointer to array containing data related to the
                                                            //!< spheres in the box
                                 int* pRawDataZ,            //!< Pointer to array containing data related to the
                                                            //!< spheres in the box
                                 int* pRawDataX_DOT,        //!< Pointer to array containing data related to
                                                            //!< the spheres in the box
                                 int* pRawDataY_DOT,        //!< Pointer to array containing data related to
                                                            //!< the spheres in the box
                                 int* pRawDataZ_DOT,        //!< Pointer to array containing data related to
                                                            //!< the spheres in the box
                                 unsigned int* SD_countsOfSpheresTouching,  //!< The array that for each
                                                                            //!< SD indicates how many
                                                                            //!< spheres touch this SD
                                 unsigned int* spheres_in_SD_composite      //!< Big array that works in conjunction
                                                                            //!< with SD_countsOfSpheresTouching.
) {
    __shared__ int sph_X[MAX_NSPHERES_PER_SD];
    __shared__ int sph_Y[MAX_NSPHERES_PER_SD];
    __shared__ int sph_Z[MAX_NSPHERES_PER_SD];
    __shared__ char bodyB_list[12 * MAX_NSPHERES_PER_SD];  // NOTE: max number of spheres that can kiss a sphere is 12.

    unsigned int spheresTouchingThisSD = SD_countsOfSpheresTouching[blockIdx.x];
    unsigned mySphereID;

    // Bring in data from global into shmem. Only a subset of threads get to do this.
    if (threadIdx.x < spheresTouchingThisSD) {
        mySphereID = spheres_in_SD_composite[blockIdx.x * MAX_NSPHERES_PER_SD + threadIdx.x];
        sph_X[threadIdx.x] = pRawDataX[mySphereID];
        sph_Y[threadIdx.x] = pRawDataY[mySphereID];
        sph_Z[threadIdx.x] = pRawDataZ[mySphereID];
    }

    __syncthreads();  // Needed to make sure data gets in shmem before using it elsewhere

    // Assumes each thread is a body, not the greatest assumption but we can fix that later
    // Note that if we have more threads than bodies, some effort gets wasted. With our current parameters (3/8/18) we
    // have at most 113 DEs per SD. If we have more bodies than threads, we might want to increase the number of threads
    // or decrease the number of DEs per SD
    unsigned int bodyA = threadIdx.x;
    double X_dir_contactForce;
    double Y_dir_contactForce;
    double Z_dir_contactForce;

    // Each body looks at each other body and computes the force that the other body exerts on it
    if (bodyA < spheresTouchingThisSD) {
        double invSphDiameter = 2. * d_monoDisperseSphRadius_SU;
        invSphDiameter = 1. / invSphDiameter;
        double X_dummyVal = sph_X[bodyA] * invSphDiameter;
        double Y_dummyVal = sph_Y[bodyA] * invSphDiameter;
        double Z_dummyVal = sph_Z[bodyA] * invSphDiameter;

        double penetrationProxy;
        unsigned int nCollisions = 0;
        for (unsigned int bodyB = 0; bodyB < spheresTouchingThisSD; bodyB++) {
            // Don't check for collision with self
            if (bodyA == bodyB)
                continue;

            // This avoids computing a square to figure our if collision or not
            X_dir_contactForce = X_dummyVal - sph_X[bodyB] * invSphDiameter;
            Y_dir_contactForce = Y_dummyVal - sph_Y[bodyB] * invSphDiameter;
            Z_dir_contactForce = Z_dummyVal - sph_Z[bodyB] * invSphDiameter;

            penetrationProxy = X_dir_contactForce * X_dir_contactForce;
            penetrationProxy += Y_dir_contactForce * Y_dir_contactForce;
            penetrationProxy += Z_dir_contactForce * Z_dir_contactForce;

            // We have a collision here...
            if (penetrationProxy < 1) {
                bodyB_list[threadIdx.x * 12 + nCollisions] = bodyB;
                nCollisions++;
            }
        }

        /**
        Compute now the forces on bodyA; i.e, what bodyA feels (if bodyA is in contact w/ anybody in this SD).






















        \f[ \mbox{penetration} = \left[\left(\frac{x_A}{2R} -
        \frac{x_B}{2R}\right)^2 + \left(\frac{y_A}{2R} - \frac{y_B}{2R}\right)^2 + \left(\frac{z_A}{2R} -
        \frac{z_B}{2R}\right)^2\right] \f]

        The deformation that enters the computation of the normal contact force is scaled by the square of the step
        size, the stiffness and the particle mass: \f[ h^2 \frac{K}{m} \delta \f] Then, the quantity that comes into
        play in computing the update in positions looks like \f[ h^2 \frac{K}{m} \times 2R \times \left( \frac{
        1}{\sqrt{(\left(\frac{x_A}{2R} - \frac{x_B}{2R}\right)^2 + \left(\frac{y_A}{2R} - \frac{y_B}{2R}\right)^2 +
        \left(\frac{z_A}{2R} - \frac{z_B}{2R}\right)^2)}} -1 \right) \times \begin{bmatrix}
        \frac{x_A}{2R} - \frac{x_B}{2R}  \vspace{0.2cm}\\
        \frac{y_A}{2R} - \frac{y_B}{2R} \vspace{0.2cm}\\
        \frac{z_A}{2R} - \frac{z_B}{2R}
        \end{bmatrix}
        \f]
        */
        float bodyA_X_velCorr = 0.f;
        float bodyA_Y_velCorr = 0.f;
        float bodyA_Z_velCorr = 0.f;
        float scalingFactor = alpha_h_bar / (psi_T_dFactor * psi_T_dFactor * psi_h_dFactor);

        for (unsigned int bodyB = 0; bodyB < nCollisions; bodyB++) {
            // Note: this can be accelerated should we decide to go w/ float. Then we can use the CUDA intrinsic:
            // __device__ ​ float rnormf ( int  dim, const float* a)
            // http://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__SINGLE.html#group__CUDA__MATH__SINGLE
            X_dir_contactForce = X_dummyVal - sph_X[bodyB] * invSphDiameter;
            Y_dir_contactForce = Y_dummyVal - sph_Y[bodyB] * invSphDiameter;
            Z_dir_contactForce = Z_dummyVal - sph_Z[bodyB] * invSphDiameter;

            penetrationProxy = X_dir_contactForce * X_dir_contactForce;
            penetrationProxy += Y_dir_contactForce * Y_dir_contactForce;
            penetrationProxy += Z_dir_contactForce * Z_dir_contactForce;

            penetrationProxy = sqrt(penetrationProxy);
            penetrationProxy = 1 / penetrationProxy;
            penetrationProxy -= 1.;

            X_dir_contactForce *= penetrationProxy;
            Y_dir_contactForce *= penetrationProxy;
            Z_dir_contactForce *= penetrationProxy;

            bodyA_X_velCorr += scalingFactor * X_dir_contactForce;
            bodyA_Y_velCorr += scalingFactor * Y_dir_contactForce;
            bodyA_Z_velCorr += scalingFactor * Z_dir_contactForce;
        }

        // Perhaps this sphere is hitting the wall[s]
        boxWallsEffects(sph_X[bodyA], sph_Y[bodyA], sph_Z[bodyA], bodyA_X_velCorr, bodyA_Y_velCorr, bodyA_Z_velCorr);

        // If the sphere belongs to this SD, add up the gravitational force component.
        // IMPORTANT: Make sure that the sphere belongs to *this* SD, otherwise we'll end up with double counting this
        // force.
        if (true) {
            // CONLAIN: can you help with this test? We add the effect of gravity more times than we need here
            bodyA_X_velCorr += alpha_h_bar * gravAcc_X_d_factor_SU;
            bodyA_Y_velCorr += alpha_h_bar * gravAcc_Y_d_factor_SU;
            bodyA_Z_velCorr += alpha_h_bar * gravAcc_Z_d_factor_SU;
        }

        // We still need to write back atomically to global memory
        atomicAdd(pRawDataX_DOT + mySphereID, (int)bodyA_X_velCorr);
        atomicAdd(pRawDataY_DOT + mySphereID, (int)bodyA_Y_velCorr);
        atomicAdd(pRawDataZ_DOT + mySphereID, (int)bodyA_Z_velCorr);
    }
}

template <unsigned int THRDS_PER_BLOCK>  //!< Number of CUB threads engaged in block-collective CUB operations.
                                         //!< Should be a multiple of 32
__global__ void updatePositions(unsigned int alpha_h_bar,  //!< The numerical integration time step
                                int* pRawDataX,            //!< Pointer to array containing data related to the
                                                           //!< spheres in the box
                                int* pRawDataY,            //!< Pointer to array containing data related to the
                                                           //!< spheres in the box
                                int* pRawDataZ,            //!< Pointer to array containing data related to the
                                                           //!< spheres in the box
                                int* pRawDataX_DOT,        //!< Pointer to array containing data related to
                                                           //!< the spheres in the box
                                int* pRawDataY_DOT,        //!< Pointer to array containing data related to
                                                           //!< the spheres in the box
                                int* pRawDataZ_DOT,        //!< Pointer to array containing data related to
                                                           //!< the spheres in the box
                                unsigned int* SD_countsOfSpheresTouching,  //!< The array that for each
                                                                           //!< SD indicates how many
                                                                           //!< spheres touch this SD
                                unsigned int* spheres_in_SD_composite,     //!< Big array that works in conjunction
                                                                           //!< with SD_countsOfSpheresTouching.
                                                                           //!< "spheres_in_SD_composite" says which
                                                                           //!< SD contains what spheres
                                unsigned int nSpheres) {
    int xSphCenter;
    int ySphCenter;
    int zSphCenter;
    // NOTE from Conlain -- somebody in this kernel is trashing heap memory and breaking things
    /// Set aside shared memory
    volatile __shared__ unsigned int offsetInComposite_SphInSD_Array[THRDS_PER_BLOCK * 8];
    volatile __shared__ bool shMem_head_flags[THRDS_PER_BLOCK * 8];

    typedef hipcub::BlockRadixSort<unsigned int, THRDS_PER_BLOCK, 8, unsigned int> BlockRadixSortOP;
    __shared__ typename BlockRadixSortOP::TempStorage temp_storage_sort;

    typedef hipcub::BlockDiscontinuity<unsigned int, THRDS_PER_BLOCK> Block_Discontinuity;
    __shared__ typename Block_Discontinuity::TempStorage temp_storage_disc;

    // Figure out what sphereID this thread will handle. We work with a 1D block structure and a 1D grid structure
    unsigned int mySphereID = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int sphIDs[8] = {mySphereID, mySphereID, mySphereID, mySphereID,
                              mySphereID, mySphereID, mySphereID, mySphereID};

    // This uses a lot of registers but is needed
    unsigned int SDsTouched[8] = {NULL_GRANULAR_ID, NULL_GRANULAR_ID, NULL_GRANULAR_ID, NULL_GRANULAR_ID,
                                  NULL_GRANULAR_ID, NULL_GRANULAR_ID, NULL_GRANULAR_ID, NULL_GRANULAR_ID};
    if (mySphereID < nSpheres) {
        // Perform numerical integration. For now, use Explicit Euler. Hitting cache, also coalesced.
        xSphCenter = alpha_h_bar * pRawDataX_DOT[mySphereID];
        ySphCenter = alpha_h_bar * pRawDataY_DOT[mySphereID];
        zSphCenter = alpha_h_bar * pRawDataZ_DOT[mySphereID];

        xSphCenter += pRawDataX[mySphereID];
        ySphCenter += pRawDataY[mySphereID];
        zSphCenter += pRawDataZ[mySphereID];

        pRawDataX[mySphereID] = xSphCenter;
        pRawDataY[mySphereID] = ySphCenter;
        pRawDataZ[mySphereID] = zSphCenter;

        figureOutTouchedSD(xSphCenter, ySphCenter, zSphCenter, SDsTouched);
    }

    __syncthreads();

    // Sort by the ID of the SD touched
    BlockRadixSortOP(temp_storage_sort).Sort(SDsTouched, sphIDs);
    __syncthreads();

    // Do a winningStreak search on whole block, might not have high utilization here
    bool head_flags[8];
    Block_Discontinuity(temp_storage_disc).FlagHeads(head_flags, SDsTouched, hipcub::Inequality());
    __syncthreads();

    // Write back to shared memory; eight-way bank conflicts here - to revisit later
    for (unsigned int i = 0; i < 8; i++) {
        shMem_head_flags[8 * threadIdx.x + i] = head_flags[i];
    }

    // Seed offsetInComposite_SphInSD_Array with "no valid ID" so that we know later on what is legit;
    // No shmem bank coflicts here, good access...
    for (unsigned int i = 0; i < 8; i++) {
        offsetInComposite_SphInSD_Array[i * THRDS_PER_BLOCK + threadIdx.x] = NULL_GRANULAR_ID;
    }

    __syncthreads();

    // Count how many times an SD shows up in conjunction with the collection of THRDS_PER_BLOCK spheres. There
    // will be some thread divergence here.
    // Loop through each potential SD, after sorting, and see if it is the start of a head
    for (unsigned int i = 0; i < 8; i++) {
        // SD currently touched, could easily be inlined
        unsigned int touchedSD = SDsTouched[i];
        if (touchedSD != NULL_GRANULAR_ID && head_flags[i]) {
            // current index into shared datastructure of length 8*THRDS_PER_BLOCK, could easily be inlined
            unsigned int idInShared = 8 * threadIdx.x + i;
            unsigned int winningStreak = 0;
            // This is the beginning of a sequence of SDs with a new ID
            do {
                winningStreak++;
                // Go until we run out of threads on the warp or until we find a new head
            } while (idInShared + winningStreak < 8 * THRDS_PER_BLOCK &&
                     !(shMem_head_flags[idInShared + winningStreak]));

            // if (touchedSD >= d_box_L_SU * d_box_D_SU * d_box_H_SU) {
            //     printf("invalid SD index %u on thread %u\n", mySphereID, touchedSD);
            // }

            // Store start of new entries
            unsigned int offset = atomicAdd(SD_countsOfSpheresTouching + touchedSD, winningStreak);

            // The value offset now gives a *relative* offset in the composite array; i.e., spheres_in_SD_composite.
            // Get the absolute offset
            offset += touchedSD * MAX_COUNT_OF_DEs_PER_SD;

            // Produce the offsets for this streak of spheres with identical SD ids
            for (unsigned int i = 0; i < winningStreak; i++)
                offsetInComposite_SphInSD_Array[idInShared + i] = offset++;
        }
    }

    __syncthreads();  // needed since we write to shared memory above; i.e., offsetInComposite_SphInSD_Array

    // Write out the data now; reister with spheres_in_SD_composite each sphere that touches a certain ID
    for (unsigned int i = 0; i < 8; i++) {
        unsigned int offset = offsetInComposite_SphInSD_Array[8 * threadIdx.x + i];
        if (offset != NULL_GRANULAR_ID)
            spheres_in_SD_composite[offset] = sphIDs[i];
    }
}

__host__ void chrono::ChGRN_MONODISP_SPH_IN_BOX_NOFRIC_SMC::copyCONSTdata_to_device() {
    // Copy quantities expressed in SU units for the SD dimensions to device
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_SD_Ldim_SU), &SD_L_SU, sizeof(d_SD_Ldim_SU)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_SD_Ddim_SU), &SD_D_SU, sizeof(d_SD_Ddim_SU)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_SD_Hdim_SU), &SD_H_SU, sizeof(d_SD_Hdim_SU)));
    // Copy global BD size in multiples of SDs to device
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_box_L_SU), &nSDs_L_SU, sizeof(d_box_L_SU)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_box_D_SU), &nSDs_D_SU, sizeof(d_box_D_SU)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_box_H_SU), &nSDs_H_SU, sizeof(d_box_H_SU)));

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(psi_T_dFactor), &psi_T_Factor, sizeof(psi_T_Factor)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(psi_h_dFactor), &psi_h_Factor, sizeof(psi_h_Factor)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(psi_L_dFactor), &psi_L_Factor, sizeof(psi_L_Factor)));

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(gravAcc_X_d_factor_SU), &gravAcc_X_factor_SU, sizeof(gravAcc_X_factor_SU)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(gravAcc_Y_d_factor_SU), &gravAcc_Y_factor_SU, sizeof(gravAcc_Y_factor_SU)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(gravAcc_Z_d_factor_SU), &gravAcc_Z_factor_SU, sizeof(gravAcc_Z_factor_SU)));

    gpuErrchk(
        hipMemcpyToSymbol(HIP_SYMBOL(d_monoDisperseSphRadius_SU), &monoDisperseSphRadius_SU, sizeof(d_monoDisperseSphRadius_SU)));
}

// Check number of spheres in each SD and dump relevant info to file
void chrono::ChGRN_MONODISP_SPH_IN_BOX_NOFRIC_SMC::checkSDCounts(std::string ofile) {
    unsigned int* sdvals = new unsigned int[nSDs];
    unsigned int* sdSpheres = new unsigned int[MAX_COUNT_OF_DEs_PER_SD * nSDs];
    unsigned int* deCounts = new unsigned int[nDEs];
    hipMemcpy(sdvals, p_device_SD_NumOf_DEs_Touching, nSDs * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipMemcpy(sdSpheres, p_device_DEs_in_SD_composite, MAX_COUNT_OF_DEs_PER_SD * nSDs * sizeof(unsigned int),
               hipMemcpyDeviceToHost);

    unsigned int max_count = 0;
    unsigned int sum = 0;
    for (unsigned int i = 0; i < nSDs; i++) {
        // printf("count is %u for SD sd %u \n", sdvals[i], i);
        sum += sdvals[i];
        if (sdvals[i] > max_count)
            max_count = sdvals[i];
    }

    printf("max DEs per SD is %u\n", max_count);
    printf("total sd/de overlaps is %u\n", sum);
    printf("theoretical total is %u\n", MAX_COUNT_OF_DEs_PER_SD * nSDs);
    // Copy over occurences in SDs
    for (unsigned int i = 0; i < MAX_COUNT_OF_DEs_PER_SD * nSDs; i++) {
        // printf("de id is %d, i is %u\n", sdSpheres[i], i);
        // Check if invalid sphere
        if (-1 == (signed int)sdSpheres[i]) {
            // printf("invalid sphere in sd");
        } else {
            deCounts[sdSpheres[i]]++;
        }
    }

    std::ofstream ptFile{ofile};
    ptFile << "x,y,z,nTouched" << std::endl;
    for (unsigned int n = 0; n < nDEs; n++) {
        ptFile << h_X_DE.at(n) << "," << h_Y_DE.at(n) << "," << h_Z_DE.at(n) << "," << deCounts[n] << std::endl;
    }
    delete[] sdvals;
    delete[] sdSpheres;
    delete[] deCounts;
}

__host__ void chrono::ChGRN_MONODISP_SPH_IN_BOX_NOFRIC_SMC::settle(float tEnd) {
    switch_to_SimUnits();
    generate_DEs();

    // Set aside memory for holding data structures worked with. Get some initializations going
    setup_simulation();
    copyCONSTdata_to_device();

    // Seed arrays that are populated by the kernel call
    const unsigned char allBitsOne = (unsigned char)-1;  // all bits of this variable are 1.
    // Set all the offsets to zero
    gpuErrchk(hipMemset(p_device_SD_NumOf_DEs_Touching, 0, nSDs * sizeof(unsigned int)));
    // For each SD, all the spheres touching that SD should have their ID be NULL_GRANULAR_ID
    gpuErrchk(
        hipMemset(p_device_DEs_in_SD_composite, allBitsOne, MAX_COUNT_OF_DEs_PER_SD * nSDs * sizeof(unsigned int)));

    /// Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nDEs + CUDA_THREADS - 1) / CUDA_THREADS;
    primingOperationsRectangularBox<CUDA_THREADS><<<nBlocks, CUDA_THREADS>>>(
        p_d_CM_X, p_d_CM_Y, p_d_CM_Z, p_device_SD_NumOf_DEs_Touching, p_device_DEs_in_SD_composite, nSpheres());
    hipDeviceSynchronize();

    // printf("checking counts\n");
    checkSDCounts("output.csv");
    // printf("counts checked\n");
    // Settling simulation loop.
    unsigned int stepSize_SU = 8;
    unsigned int tEnd_SU = tEnd / TIME_UNIT;
    for (unsigned int crntTime_SU = 0; crntTime_SU < tEnd; crntTime_SU += stepSize_SU) {
        updateVelocities<MAX_COUNT_OF_DEs_PER_SD><<<nSDs, MAX_COUNT_OF_DEs_PER_SD>>>(
            stepSize_SU, p_d_CM_X, p_d_CM_Y, p_d_CM_Z, p_d_CM_XDOT, p_d_CM_XDOT, p_d_CM_XDOT,
            p_device_SD_NumOf_DEs_Touching, p_device_DEs_in_SD_composite);
        hipDeviceSynchronize();
        gpuErrchk(hipMemset(p_device_SD_NumOf_DEs_Touching, 0, nSDs * sizeof(unsigned int)));
        gpuErrchk(hipMemset(p_device_DEs_in_SD_composite, allBitsOne,
                             MAX_COUNT_OF_DEs_PER_SD * nSDs * sizeof(unsigned int)));
        hipDeviceSynchronize();

        updatePositions<CUDA_THREADS><<<nBlocks, CUDA_THREADS>>>(
            stepSize_SU, p_d_CM_X, p_d_CM_Y, p_d_CM_Z, p_d_CM_XDOT, p_d_CM_XDOT, p_d_CM_XDOT,
            p_device_SD_NumOf_DEs_Touching, p_device_DEs_in_SD_composite, nSpheres());
        hipDeviceSynchronize();
    }

    cleanup_simulation();
    return;
}
