#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2018 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Dan Negrut, Conlain Kelly, Nic Olsen
// =============================================================================

#include "chrono_granular/physics/ChGranularGPU_SMC.cuh"

namespace chrono {
namespace granular {

__host__ double ChSystemGranular_MonodisperseSMC::get_max_z() const {
    int* max_z_d;
    int max_z_h;
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    gpuErrchk(hipMalloc(&max_z_d, sizeof(int)));

    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, pos_Z.data(), max_z_d, nDEs);
    gpuErrchk(hipDeviceSynchronize());

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run max-reduction
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, pos_Z.data(), max_z_d, nDEs);
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(&max_z_h, max_z_d, sizeof(int), hipMemcpyDeviceToHost));

    double max_z_UU = max_z_h * gran_params->LENGTH_UNIT;
    gpuErrchk(hipFree(max_z_d));
    gpuErrchk(hipDeviceSynchronize());

    return max_z_UU;
}

/// Copy constant sphere data to device, this should run at start
__host__ void ChSystemGranular_MonodisperseSMC::copyConstSphereDataToDevice() {
    // Copy quantities expressed in SU units for the SD dimensions to device
    gran_params->SD_size_X_SU = SD_size_X_SU;
    gran_params->SD_size_Y_SU = SD_size_Y_SU;
    gran_params->SD_size_Z_SU = SD_size_Z_SU;
    // Copy global BD size in multiples of SDs to device
    gran_params->nSDs_X = nSDs_X;
    gran_params->nSDs_Y = nSDs_Y;
    gran_params->nSDs_Z = nSDs_Z;

    gran_params->max_x_pos_unsigned = ((int64_t)gran_params->SD_size_X_SU * gran_params->nSDs_X);
    gran_params->max_y_pos_unsigned = ((int64_t)gran_params->SD_size_Y_SU * gran_params->nSDs_Y);
    gran_params->max_z_pos_unsigned = ((int64_t)gran_params->SD_size_Z_SU * gran_params->nSDs_Z);

    printf("max pos is is %lu, %lu, %lu\n", gran_params->max_x_pos_unsigned, gran_params->max_y_pos_unsigned,
           gran_params->max_z_pos_unsigned);

    gran_params->gravAcc_X_SU = gravity_X_SU;
    gran_params->gravAcc_Y_SU = gravity_Y_SU;
    gran_params->gravAcc_Z_SU = gravity_Z_SU;
    gran_params->gravMag_SU =
        std::sqrt(gravity_X_SU * gravity_X_SU + gravity_Y_SU * gravity_Y_SU + gravity_Z_SU * gravity_Z_SU);

    gran_params->sphereRadius_SU = sphereRadius_SU;
    // NOTE: Assumes mass = 1
    gran_params->sphereInertia_by_r = 2.f / 5.f * gran_params->sphere_mass_SU * sphereRadius_SU;

    gran_params->K_n_s2s_SU = K_n_s2s_SU;
    gran_params->K_n_s2w_SU = K_n_s2w_SU;
    gran_params->Gamma_n_s2s_SU = Gamma_n_s2s_SU;
    gran_params->Gamma_n_s2w_SU = Gamma_n_s2w_SU;

    gran_params->K_t_s2s_SU = K_t_s2s_SU;
    gran_params->K_t_s2w_SU = K_t_s2w_SU;
    gran_params->Gamma_t_s2s_SU = Gamma_t_s2s_SU;
    gran_params->Gamma_t_s2w_SU = Gamma_t_s2w_SU;

    gran_params->cohesion_ratio = cohesion_over_gravity;
    gran_params->adhesion_ratio_s2w = adhesion_s2w_over_gravity;

    gran_params->integrator_type = time_integrator;
    gran_params->contact_model = contact_model;
    gran_params->friction_mode = fric_mode;
}

/// Similar to the copyConstSphereDataToDevice, but saves us a big copy
/// This can run at every timestep to allow a moving BD
__host__ void ChSystemGranular_MonodisperseSMC::copyBDFrameToDevice() {
    // Unified memory does all the work here
    gran_params->BD_frame_X = BD_frame_X;
    gran_params->BD_frame_Y = BD_frame_Y;
    gran_params->BD_frame_Z = BD_frame_Z;
    gran_params->BD_frame_X_dot = BD_frame_X_dot;
    gran_params->BD_frame_Y_dot = BD_frame_Y_dot;
    gran_params->BD_frame_Z_dot = BD_frame_Z_dot;
    gpuErrchk(hipDeviceSynchronize());
}

// Check number of spheres in each SD and dump relevant info to file
void ChSystemGranular_MonodisperseSMC::checkSDCounts(std::string ofile, bool write_out = false, bool verbose = false) {
    // Count of DEs in each SD
    unsigned int* sdvals = SD_NumOf_DEs_Touching.data();
    // DEs that are in each SD
    unsigned int* sdSpheres = DEs_in_SD_composite.data();
    // # times each DE appears in some SD
    unsigned int* deCounts = new unsigned int[nDEs];

    // could use memset instead, just need to zero these out
    for (unsigned int i = 0; i < nDEs; i++) {
        deCounts[i] = 0;
    }

    unsigned int max_count = 0;
    unsigned int sum = 0;
    for (unsigned int i = 0; i < nSDs; i++) {
        // printf("count is %u for SD sd %u \n", sdvals[i], i);
        sum += sdvals[i];
        if (sdvals[i] > max_count)
            max_count = sdvals[i];
    }
    // safety checks, if these fail we were probably about to crash
    assert(sum < MAX_COUNT_OF_DEs_PER_SD * nSDs);
    assert(max_count < MAX_COUNT_OF_DEs_PER_SD);
    if (verbose) {
        printf("max DEs per SD is %u\n", max_count);
        printf("total sd/de overlaps is %u\n", sum);
        printf("theoretical total is %u\n", MAX_COUNT_OF_DEs_PER_SD * nSDs);
    }
    // Copy over occurences in SDs
    for (unsigned int i = 0; i < MAX_COUNT_OF_DEs_PER_SD * nSDs; i++) {
        // printf("de id is %d, i is %u\n", sdSpheres[i], i);
        // Check if invalid sphere
        if (sdSpheres[i] == NULL_GRANULAR_ID) {
            // printf("invalid sphere in sd");
        } else {
            assert(sdSpheres[i] < nDEs);
            deCounts[sdSpheres[i]]++;
        }
    }
    if (write_out) {
        writeFile(ofile, deCounts);
    }
    delete[] deCounts;
}
// This can belong to the superclass but does reference deCounts which may not be a thing when DVI rolls around
void ChSystemGranular_MonodisperseSMC::writeFile(std::string ofile, unsigned int* deCounts) {
    // unnecessary if called by checkSDCounts()
    // The file writes are a pretty big slowdown in CSV mode
    if (file_write_mode == GRAN_OUTPUT_MODE::BINARY) {
        // Write the data as binary to a file, requires later postprocessing that can be done in parallel, this is a
        // much faster write due to no formatting
        std::ofstream ptFile(ofile + ".raw", std::ios::out | std::ios::binary);

        for (unsigned int n = 0; n < nDEs; n++) {
            float absv = sqrt(pos_X_dt.at(n) * pos_X_dt.at(n) + pos_Y_dt.at(n) * pos_Y_dt.at(n) +
                              pos_Z_dt.at(n) * pos_Z_dt.at(n));

            ptFile.write((const char*)&pos_X.at(n), sizeof(int));
            ptFile.write((const char*)&pos_Y.at(n), sizeof(int));
            ptFile.write((const char*)&pos_Z.at(n), sizeof(int));
            ptFile.write((const char*)&pos_X_dt.at(n), sizeof(float));
            ptFile.write((const char*)&pos_Y_dt.at(n), sizeof(float));
            ptFile.write((const char*)&pos_Z_dt.at(n), sizeof(float));
            ptFile.write((const char*)&absv, sizeof(float));
            ptFile.write((const char*)&deCounts[n], sizeof(int));
        }
    } else if (file_write_mode == GRAN_OUTPUT_MODE::CSV) {
        // CSV is much slower but requires less postprocessing
        std::ofstream ptFile(ofile + ".csv", std::ios::out);

        // Dump to a stream, write to file only at end
        std::ostringstream outstrstream;
        outstrstream << "x,y,z,vx,vy,vz,absv,nTouched\n";

        for (unsigned int n = 0; n < nDEs; n++) {
            float absv = sqrt(pos_X_dt.at(n) * pos_X_dt.at(n) + pos_Y_dt.at(n) * pos_Y_dt.at(n) +
                              pos_Z_dt.at(n) * pos_Z_dt.at(n));
            outstrstream << pos_X.at(n) << "," << pos_Y.at(n) << "," << pos_Z.at(n) << "," << pos_X_dt.at(n) << ","
                         << pos_Y_dt.at(n) << "," << pos_Z_dt.at(n) << "," << absv << "," << deCounts[n] << "\n";
        }

        ptFile << outstrstream.str();
    } else if (file_write_mode == GRAN_OUTPUT_MODE::NONE) {
        // Do nothing, only here for symmetry
    }
}

// This can belong to the superclass but does reference deCounts which may not be a thing when DVI rolls around
void ChSystemGranular_MonodisperseSMC::writeFileUU(std::string ofile) {
    // The file writes are a pretty big slowdown in CSV mode
    if (file_write_mode == GRAN_OUTPUT_MODE::BINARY) {
        // TODO implement this
        // Write the data as binary to a file, requires later postprocessing that can be done in parallel, this is a
        // much faster write due to no formatting
        std::ofstream ptFile(ofile + ".raw", std::ios::out | std::ios::binary);

        for (unsigned int n = 0; n < nDEs; n++) {
            float absv = sqrt(pos_X_dt.at(n) * pos_X_dt.at(n) + pos_Y_dt.at(n) * pos_Y_dt.at(n) +
                              pos_Z_dt.at(n) * pos_Z_dt.at(n)) *
                         (gran_params->LENGTH_UNIT / gran_params->TIME_UNIT);
            float x_UU = pos_X[n] * gran_params->LENGTH_UNIT;
            float y_UU = pos_Y[n] * gran_params->LENGTH_UNIT;
            float z_UU = pos_Z[n] * gran_params->LENGTH_UNIT;

            ptFile.write((const char*)&x_UU, sizeof(float));
            ptFile.write((const char*)&y_UU, sizeof(float));
            ptFile.write((const char*)&z_UU, sizeof(float));
            ptFile.write((const char*)&absv, sizeof(float));

            if (fric_mode != GRAN_FRICTION_MODE::FRICTIONLESS) {
                ptFile.write((const char*)&sphere_Omega_X.at(n), sizeof(float));
                ptFile.write((const char*)&sphere_Omega_Y.at(n), sizeof(float));
                ptFile.write((const char*)&sphere_Omega_Z.at(n), sizeof(float));
            }
        }
    } else if (file_write_mode == GRAN_OUTPUT_MODE::CSV) {
        // CSV is much slower but requires less postprocessing
        std::ofstream ptFile(ofile + ".csv", std::ios::out);

        // Dump to a stream, write to file only at end
        std::ostringstream outstrstream;
        outstrstream << "x,y,z,absv";

        if (fric_mode != GRAN_FRICTION_MODE::FRICTIONLESS) {
            outstrstream << ",wx,wy,wz";
        }
        outstrstream << "\n";
        for (unsigned int n = 0; n < nDEs; n++) {
            float absv = sqrt(pos_X_dt.at(n) * pos_X_dt.at(n) + pos_Y_dt.at(n) * pos_Y_dt.at(n) +
                              pos_Z_dt.at(n) * pos_Z_dt.at(n)) *
                         (gran_params->LENGTH_UNIT / gran_params->TIME_UNIT);
            float x_UU = pos_X[n] * gran_params->LENGTH_UNIT;
            float y_UU = pos_Y[n] * gran_params->LENGTH_UNIT;
            float z_UU = pos_Z[n] * gran_params->LENGTH_UNIT;

            outstrstream << x_UU << "," << y_UU << "," << z_UU << "," << absv;

            if (fric_mode != GRAN_FRICTION_MODE::FRICTIONLESS) {
                outstrstream << "," << sphere_Omega_X.at(n) << "," << sphere_Omega_Y.at(n) << ","
                             << sphere_Omega_Z.at(n);
            }
            outstrstream << "\n";
        }

        ptFile << outstrstream.str();
    } else if (file_write_mode == GRAN_OUTPUT_MODE::NONE) {
        // Do nothing, only here for symmetry
    }
}

// Reset broadphase data structures
void ChSystemGranular_MonodisperseSMC::resetBroadphaseInformation() {
    // Set all the offsets to zero
    gpuErrchk(hipMemset(SD_NumOf_DEs_Touching.data(), 0, nSDs * sizeof(unsigned int)));
    // For each SD, all the spheres touching that SD should have their ID be NULL_GRANULAR_ID
    gpuErrchk(hipMemset(DEs_in_SD_composite.data(), NULL_GRANULAR_ID,
                         MAX_COUNT_OF_DEs_PER_SD * nSDs * sizeof(unsigned int)));
}
// Reset sphere-sphere force data structures
void ChSystemGranular_MonodisperseSMC::resetSphereForces() {
    // cache past force data
    if (time_integrator == GRAN_TIME_INTEGRATOR::CHUNG) {
        gpuErrchk(hipMemcpy(sphere_force_X_old.data(), sphere_force_X.data(), nDEs * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(sphere_force_Y_old.data(), sphere_force_Y.data(), nDEs * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(sphere_force_Z_old.data(), sphere_force_Z.data(), nDEs * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipDeviceSynchronize());
    }
    // reset forces to zero
    gpuErrchk(hipMemset(sphere_force_X.data(), 0, nDEs * sizeof(float)));
    gpuErrchk(hipMemset(sphere_force_Y.data(), 0, nDEs * sizeof(float)));
    gpuErrchk(hipMemset(sphere_force_Z.data(), 0, nDEs * sizeof(float)));

    // reset torques to zero, if applicable
    if (fric_mode != FRICTIONLESS) {
        gpuErrchk(hipMemset(sphere_ang_acc_X.data(), 0, nDEs * sizeof(float)));
        gpuErrchk(hipMemset(sphere_ang_acc_Y.data(), 0, nDEs * sizeof(float)));
        gpuErrchk(hipMemset(sphere_ang_acc_Z.data(), 0, nDEs * sizeof(float)));
    }
}

void ChSystemGranular_MonodisperseSMC::updateBDPosition(const float stepSize_SU) {
    // Frequency of oscillation
    float frame_X_old = BD_frame_X;
    float frame_Y_old = BD_frame_Y;
    float frame_Z_old = BD_frame_Z;
    // Put the bottom-left corner of box wherever the user told us to
    BD_frame_X = (box_size_X * (BDPositionFunctionX(elapsedSimTime))) / gran_params->LENGTH_UNIT;
    BD_frame_Y = (box_size_Y * (BDPositionFunctionY(elapsedSimTime))) / gran_params->LENGTH_UNIT;
    BD_frame_Z = (box_size_Z * (BDPositionFunctionZ(elapsedSimTime))) / gran_params->LENGTH_UNIT;

    BD_frame_X_dot = (BD_frame_X - frame_X_old) / stepSize_SU;
    BD_frame_Y_dot = (BD_frame_Y - frame_Y_old) / stepSize_SU;
    BD_frame_Z_dot = (BD_frame_Z - frame_Z_old) / stepSize_SU;

    copyBDFrameToDevice();
}

// All the information a moving sphere needs
typedef struct {
    int pos_X;
    int pos_Y;
    int pos_Z;
    float pos_X_dt;
    float pos_Y_dt;
    float pos_Z_dt;
} sphere_data_struct;

template <unsigned int CUB_THREADS>
__global__ void owner_prepack(int* d_sphere_pos_X,
                              int* d_sphere_pos_Y,
                              int* d_sphere_pos_Z,
                              float* d_sphere_pos_X_dt,
                              float* d_sphere_pos_Y_dt,
                              float* d_sphere_pos_Z_dt,
                              unsigned int nSpheres,
                              unsigned int* owners,
                              sphere_data_struct* sphere_info,
                              GranParamsPtr gran_params) {
    // Figure out what sphereID this thread will handle. We work with a 1D block structure and a 1D grid structure
    unsigned int mySphereID = threadIdx.x + blockIdx.x * blockDim.x;
    // Only do this for valid spheres
    if (mySphereID >= nSpheres) {
        return;
    }
    // Find this SD's owner
    owners[mySphereID] = SDTripletID(
        pointSDTriplet(d_sphere_pos_X[mySphereID], d_sphere_pos_Y[mySphereID], d_sphere_pos_Z[mySphereID], gran_params),
        gran_params);

    sphere_data_struct mydata = sphere_info[mySphereID];

    // The value is the sphere id, to be sorted with owner as the key
    mydata.pos_X = d_sphere_pos_X[mySphereID];
    mydata.pos_Y = d_sphere_pos_Y[mySphereID];
    mydata.pos_Z = d_sphere_pos_Z[mySphereID];
    mydata.pos_X_dt = d_sphere_pos_X_dt[mySphereID];
    mydata.pos_Y_dt = d_sphere_pos_Y_dt[mySphereID];
    mydata.pos_Z_dt = d_sphere_pos_Z_dt[mySphereID];
    // replace with the new data
    sphere_info[mySphereID] = mydata;
}
// unpack sorted data to global memory, very coalesced
template <unsigned int CUB_THREADS>
__global__ void owner_unpack(int* d_sphere_pos_X,
                             int* d_sphere_pos_Y,
                             int* d_sphere_pos_Z,
                             float* d_sphere_pos_X_dt,
                             float* d_sphere_pos_Y_dt,
                             float* d_sphere_pos_Z_dt,
                             unsigned int nSpheres,
                             sphere_data_struct* sphere_info,
                             GranParamsPtr gran_params) {
    // Figure out what sphereID this thread will handle. We work with a 1D block structure and a 1D grid structure
    unsigned int mySphereID = threadIdx.x + blockIdx.x * blockDim.x;
    // Only do this for valid spheres
    if (mySphereID >= nSpheres) {
        return;
    }
    sphere_data_struct mydata = sphere_info[mySphereID];

    // The value is the sphere id, to be sorted with owner as the key
    d_sphere_pos_X[mySphereID] = mydata.pos_X;
    d_sphere_pos_Y[mySphereID] = mydata.pos_Y;
    d_sphere_pos_Z[mySphereID] = mydata.pos_Z;
    d_sphere_pos_X_dt[mySphereID] = mydata.pos_X_dt;
    d_sphere_pos_Y_dt[mySphereID] = mydata.pos_Y_dt;
    d_sphere_pos_Z_dt[mySphereID] = mydata.pos_Z_dt;
}

// Sorts data by owner SD, makes nicer memory accesses
// Uses a boatload of memory
__host__ void ChSystemGranular_MonodisperseSMC::defragment_data() {
    VERBOSE_PRINTF("Starting defrag run!\n");
    unsigned int nBlocks = (nDEs + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    // Set of pointers for each buffer
    unsigned int* d_owners;
    sphere_data_struct* d_sphere_data;
    // second buffer for nice sort
    unsigned int* d_owners_2;
    sphere_data_struct* d_sphere_data_2;
    // Allocate some nice memory
    gpuErrchk(hipMalloc(&d_owners, nDEs * sizeof(unsigned int)));
    gpuErrchk(hipMalloc(&d_sphere_data, nDEs * sizeof(sphere_data_struct)));
    gpuErrchk(hipMalloc(&d_owners_2, nDEs * sizeof(unsigned int)));
    gpuErrchk(hipMalloc(&d_sphere_data_2, nDEs * sizeof(sphere_data_struct)));
    owner_prepack<CUDA_THREADS_PER_BLOCK><<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(
        pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data(), nDEs, d_owners,
        d_sphere_data, gran_params);
    gpuErrchk(hipDeviceSynchronize());

    // Create a set of DoubleBuffers to wrap pairs of device pointers
    hipcub::DoubleBuffer<unsigned int> d_keys(d_owners, d_owners_2);
    hipcub::DoubleBuffer<sphere_data_struct> d_values(d_sphere_data, d_sphere_data_2);

    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    // pass null, cub tells us what it needs
    hipcub::DeviceRadixSort::SortPairs(NULL, temp_storage_bytes, d_keys, d_values, nDEs);
    gpuErrchk(hipDeviceSynchronize());

    // Allocate temporary storage
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    gpuErrchk(hipDeviceSynchronize());

    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, nDEs);
    gpuErrchk(hipDeviceSynchronize());

    owner_unpack<CUDA_THREADS_PER_BLOCK>
        <<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt.data(),
                                              pos_Y_dt.data(), pos_Z_dt.data(), nDEs, d_values.Current(), gran_params);
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_owners);
    hipFree(d_owners_2);
    hipFree(d_sphere_data);
    hipFree(d_sphere_data_2);
    hipFree(d_temp_storage);
    VERBOSE_PRINTF("defrag finished!\n");
}

__global__ void generate_absv(const unsigned int nDEs,
                              const float* velX,
                              const float* velY,
                              const float* velZ,
                              float* d_absv) {
    unsigned int my_sphere = blockIdx.x * blockDim.x + threadIdx.x;
    if (my_sphere < nDEs) {
        float v[3] = {velX[my_sphere], velY[my_sphere], velZ[my_sphere]};
        d_absv[my_sphere] = v[0] * v[0] + v[1] * v[1] + v[2] * v[2];
    }
}

__host__ float ChSystemGranular_MonodisperseSMC::get_max_vel() {
    float* d_absv;
    float* d_max_vel;
    float h_max_vel;
    gpuErrchk(hipMalloc(&d_absv, nDEs * sizeof(float)));
    gpuErrchk(hipMalloc(&d_max_vel, sizeof(float)));

    generate_absv<<<(nDEs + 255) / 256, 256>>>(nDEs, pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data(), d_absv);

    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_absv, d_max_vel, nDEs);
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_absv, d_max_vel, nDEs);
    gpuErrchk(hipMemcpy(&h_max_vel, d_max_vel, sizeof(float), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_absv));
    gpuErrchk(hipFree(d_max_vel));

    return h_max_vel;
}

__host__ void ChSystemGranular_MonodisperseSMC::runInitialSpherePriming() {
    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nDEs + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
    printf("doing priming!\n");
    printf("max possible composite offset is %zu\n", (size_t)nSDs * MAX_COUNT_OF_DEs_PER_SD);
    sphereDataStruct sphere_data;

    packSphereDataPointers(sphere_data);

    primingOperationsRectangularBox<CUDA_THREADS_PER_BLOCK>
        <<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data, nDEs, gran_params);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
    printf("priming finished!\n");
}

__host__ double ChSystemGranular_MonodisperseSMC::advance_simulation(float duration) {
    sphereDataStruct sphere_data;
    packSphereDataPointers(sphere_data);

    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nDEs + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    // Settling simulation loop.
    float duration_SU = std::ceil(duration / gran_params->TIME_UNIT);
    unsigned int nsteps = duration_SU / stepSize_SU;

    VERBOSE_PRINTF("advancing by %f at timestep %f, %u timesteps at approx user timestep %f\n", duration_SU,
                   stepSize_SU, nsteps, duration / nsteps);
    float time_elapsed_SU = 0;  // time elapsed in this advance call
    // Run the simulation, there are aggressive synchronizations because we want to have no race conditions
    for (; time_elapsed_SU < stepSize_SU * nsteps; time_elapsed_SU += stepSize_SU) {
        determineNewStepSize_SU();  // doesn't always change the timestep

        gran_params->alpha_h_bar = stepSize_SU;
        // Update the position and velocity of the BD, if relevant
        if (!BD_is_fixed) {
            updateBDPosition(stepSize_SU);  // TODO current time
        }
        resetSphereForces();

        VERBOSE_PRINTF("Starting computeSphereForces!\n");

        // Compute sphere-sphere forces
        computeSphereForces<MAX_COUNT_OF_DEs_PER_SD><<<nSDs, MAX_COUNT_OF_DEs_PER_SD>>>(
            sphere_data, gran_params, BC_type_list.data(), BC_params_list_SU.data(), BC_params_list_SU.size());

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        VERBOSE_PRINTF("Resetting broadphase info!\n");

        resetBroadphaseInformation();

        VERBOSE_PRINTF("Starting updatePositions!\n");
        updatePositions<CUDA_THREADS_PER_BLOCK>
            <<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(stepSize_SU, sphere_data, nDEs, gran_params);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        elapsedSimTime += stepSize_SU * gran_params->TIME_UNIT;  // Advance current time
    }

    return time_elapsed_SU * gran_params->TIME_UNIT;  // return elapsed UU time
}
}  // namespace granular
}  // namespace chrono
