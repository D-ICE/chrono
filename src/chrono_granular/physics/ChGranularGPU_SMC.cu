#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2018 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Dan Negrut, Conlain Kelly, Nic Olsen
// =============================================================================

#include "chrono_granular/physics/ChGranularGPU_SMC.cuh"

namespace chrono {
namespace granular {

__host__ double ChSystemGranular_MonodisperseSMC::get_max_z() const {
    int* max_z_d;
    int max_z_h;
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    gpuErrchk(hipMalloc(&max_z_d, sizeof(int)));

    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, pos_Z.data(), max_z_d, nSpheres);
    gpuErrchk(hipDeviceSynchronize());

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run max-reduction
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, pos_Z.data(), max_z_d, nSpheres);
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(&max_z_h, max_z_d, sizeof(int), hipMemcpyDeviceToHost));

    double max_z_UU = max_z_h * gran_params->LENGTH_UNIT;
    gpuErrchk(hipFree(max_z_d));
    gpuErrchk(hipDeviceSynchronize());

    return max_z_UU;
}

// Reset broadphase data structures
void ChSystemGranular_MonodisperseSMC::resetBroadphaseInformation() {
    // Set all the offsets to zero
    gpuErrchk(hipMemset(SD_NumSpheresTouching.data(), 0, SD_NumSpheresTouching.size() * sizeof(unsigned int)));
    gpuErrchk(hipMemset(SD_SphereCompositeOffsets.data(), 0, SD_SphereCompositeOffsets.size() * sizeof(unsigned int)));
    // For each SD, all the spheres touching that SD should have their ID be NULL_GRANULAR_ID
    gpuErrchk(hipMemset(spheres_in_SD_composite.data(), NULL_GRANULAR_ID,
                         spheres_in_SD_composite.size() * sizeof(unsigned int)));
    gpuErrchk(hipDeviceSynchronize());
}

// Reset sphere acceleration data structures
void ChSystemGranular_MonodisperseSMC::resetSphereAccelerations() {
    // cache past acceleration data
    if (time_integrator == GRAN_TIME_INTEGRATOR::CHUNG || time_integrator == GRAN_TIME_INTEGRATOR::VELOCITY_VERLET) {
        gpuErrchk(hipMemcpy(sphere_acc_X_old.data(), sphere_acc_X.data(), nSpheres * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(sphere_acc_Y_old.data(), sphere_acc_Y.data(), nSpheres * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(sphere_acc_Z_old.data(), sphere_acc_Z.data(), nSpheres * sizeof(float),
                             hipMemcpyDeviceToDevice));
        // if we have multistep AND friction, cache old alphas
        if (gran_params->friction_mode != FRICTIONLESS) {
            gpuErrchk(hipMemcpy(sphere_ang_acc_X_old.data(), sphere_ang_acc_X.data(), nSpheres * sizeof(float),
                                 hipMemcpyDeviceToDevice));
            gpuErrchk(hipMemcpy(sphere_ang_acc_Y_old.data(), sphere_ang_acc_Y.data(), nSpheres * sizeof(float),
                                 hipMemcpyDeviceToDevice));
            gpuErrchk(hipMemcpy(sphere_ang_acc_Z_old.data(), sphere_ang_acc_Z.data(), nSpheres * sizeof(float),
                                 hipMemcpyDeviceToDevice));
        }
        gpuErrchk(hipDeviceSynchronize());
    }

    // reset current accelerations to zero to zero
    gpuErrchk(hipMemset(sphere_acc_X.data(), 0, nSpheres * sizeof(float)));
    gpuErrchk(hipMemset(sphere_acc_Y.data(), 0, nSpheres * sizeof(float)));
    gpuErrchk(hipMemset(sphere_acc_Z.data(), 0, nSpheres * sizeof(float)));

    // reset torques to zero, if applicable
    if (gran_params->friction_mode != FRICTIONLESS) {
        gpuErrchk(hipMemset(sphere_ang_acc_X.data(), 0, nSpheres * sizeof(float)));
        gpuErrchk(hipMemset(sphere_ang_acc_Y.data(), 0, nSpheres * sizeof(float)));
        gpuErrchk(hipMemset(sphere_ang_acc_Z.data(), 0, nSpheres * sizeof(float)));
    }
}

// All the information a moving sphere needs
typedef struct {
    int pos_X;
    int pos_Y;
    int pos_Z;
    float pos_X_dt;
    float pos_Y_dt;
    float pos_Z_dt;
} sphere_data_struct;

template <unsigned int CUB_THREADS>
__global__ void owner_prepack(int* d_sphere_pos_X,
                              int* d_sphere_pos_Y,
                              int* d_sphere_pos_Z,
                              float* d_sphere_pos_X_dt,
                              float* d_sphere_pos_Y_dt,
                              float* d_sphere_pos_Z_dt,
                              unsigned int nSpheres,
                              unsigned int* owners,
                              sphere_data_struct* sphere_info,
                              GranParamsPtr gran_params) {
    // Figure out what sphereID this thread will handle. We work with a 1D block structure and a 1D grid structure
    unsigned int mySphereID = threadIdx.x + blockIdx.x * blockDim.x;
    // Only do this for valid spheres
    if (mySphereID >= nSpheres) {
        return;
    }
    // Find this SD's owner
    owners[mySphereID] = SDTripletID(
        pointSDTriplet(d_sphere_pos_X[mySphereID], d_sphere_pos_Y[mySphereID], d_sphere_pos_Z[mySphereID], gran_params),
        gran_params);

    sphere_data_struct mydata = sphere_info[mySphereID];

    // The value is the sphere id, to be sorted with owner as the key
    mydata.pos_X = d_sphere_pos_X[mySphereID];
    mydata.pos_Y = d_sphere_pos_Y[mySphereID];
    mydata.pos_Z = d_sphere_pos_Z[mySphereID];
    mydata.pos_X_dt = d_sphere_pos_X_dt[mySphereID];
    mydata.pos_Y_dt = d_sphere_pos_Y_dt[mySphereID];
    mydata.pos_Z_dt = d_sphere_pos_Z_dt[mySphereID];
    // replace with the new data
    sphere_info[mySphereID] = mydata;
}
// unpack sorted data to global memory, very coalesced
template <unsigned int CUB_THREADS>
__global__ void owner_unpack(int* d_sphere_pos_X,
                             int* d_sphere_pos_Y,
                             int* d_sphere_pos_Z,
                             float* d_sphere_pos_X_dt,
                             float* d_sphere_pos_Y_dt,
                             float* d_sphere_pos_Z_dt,
                             unsigned int nSpheres,
                             sphere_data_struct* sphere_info,
                             GranParamsPtr gran_params) {
    // Figure out what sphereID this thread will handle. We work with a 1D block structure and a 1D grid structure
    unsigned int mySphereID = threadIdx.x + blockIdx.x * blockDim.x;
    // Only do this for valid spheres
    if (mySphereID >= nSpheres) {
        return;
    }
    sphere_data_struct mydata = sphere_info[mySphereID];

    // The value is the sphere id, to be sorted with owner as the key
    d_sphere_pos_X[mySphereID] = mydata.pos_X;
    d_sphere_pos_Y[mySphereID] = mydata.pos_Y;
    d_sphere_pos_Z[mySphereID] = mydata.pos_Z;
    d_sphere_pos_X_dt[mySphereID] = mydata.pos_X_dt;
    d_sphere_pos_Y_dt[mySphereID] = mydata.pos_Y_dt;
    d_sphere_pos_Z_dt[mySphereID] = mydata.pos_Z_dt;
}

// Sorts data by owner SD, makes nicer memory accesses
// Uses a boatload of memory
__host__ void ChSystemGranular_MonodisperseSMC::defragment_data() {
    VERBOSE_PRINTF("Starting defrag run!\n");
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    // Set of pointers for each buffer
    unsigned int* d_owners;
    sphere_data_struct* d_sphere_data;
    // second buffer for nice sort
    unsigned int* d_owners_2;
    sphere_data_struct* d_sphere_data_2;
    // Allocate some nice memory
    gpuErrchk(hipMalloc(&d_owners, nSpheres * sizeof(unsigned int)));
    gpuErrchk(hipMalloc(&d_sphere_data, nSpheres * sizeof(sphere_data_struct)));
    gpuErrchk(hipMalloc(&d_owners_2, nSpheres * sizeof(unsigned int)));
    gpuErrchk(hipMalloc(&d_sphere_data_2, nSpheres * sizeof(sphere_data_struct)));
    owner_prepack<CUDA_THREADS_PER_BLOCK><<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(
        pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data(), nSpheres, d_owners,
        d_sphere_data, gran_params);
    gpuErrchk(hipDeviceSynchronize());

    // Create a set of DoubleBuffers to wrap pairs of device pointers
    hipcub::DoubleBuffer<unsigned int> d_keys(d_owners, d_owners_2);
    hipcub::DoubleBuffer<sphere_data_struct> d_values(d_sphere_data, d_sphere_data_2);

    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    // pass null, cub tells us what it needs
    hipcub::DeviceRadixSort::SortPairs(NULL, temp_storage_bytes, d_keys, d_values, nSpheres);
    gpuErrchk(hipDeviceSynchronize());

    // Allocate temporary storage
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    gpuErrchk(hipDeviceSynchronize());

    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, nSpheres);
    gpuErrchk(hipDeviceSynchronize());

    owner_unpack<CUDA_THREADS_PER_BLOCK><<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(
        pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data(), nSpheres,
        d_values.Current(), gran_params);
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_owners);
    hipFree(d_owners_2);
    hipFree(d_sphere_data);
    hipFree(d_sphere_data_2);
    hipFree(d_temp_storage);
    VERBOSE_PRINTF("defrag finished!\n");
}

__global__ void generate_absv(const unsigned int nSpheres,
                              const float* velX,
                              const float* velY,
                              const float* velZ,
                              float* d_absv) {
    unsigned int my_sphere = blockIdx.x * blockDim.x + threadIdx.x;
    if (my_sphere < nSpheres) {
        float v[3] = {velX[my_sphere], velY[my_sphere], velZ[my_sphere]};
        d_absv[my_sphere] = v[0] * v[0] + v[1] * v[1] + v[2] * v[2];
    }
}

__host__ float ChSystemGranular_MonodisperseSMC::get_max_vel() const {
    float* d_absv;
    float* d_max_vel;
    float h_max_vel;
    gpuErrchk(hipMalloc(&d_absv, nSpheres * sizeof(float)));
    gpuErrchk(hipMalloc(&d_max_vel, sizeof(float)));

    generate_absv<<<(nSpheres + 255) / 256, 256>>>(nSpheres, pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data(), d_absv);

    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_absv, d_max_vel, nSpheres);
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_absv, d_max_vel, nSpheres);
    gpuErrchk(hipMemcpy(&h_max_vel, d_max_vel, sizeof(float), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_absv));
    gpuErrchk(hipFree(d_max_vel));

    return h_max_vel;
}

__host__ void ChSystemGranular_MonodisperseSMC::runSphereBroadphase() {
    VERBOSE_PRINTF("Resetting broadphase info!\n");

    resetBroadphaseInformation();
    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    sphereDataStruct sphere_data;

    packSphereDataPointers(sphere_data);

    sphereBroadphase_dryrun<CUDA_THREADS_PER_BLOCK>
        <<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data, nSpheres, gran_params);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    // num spheres in last SD
    unsigned int last_SD_num_spheres = SD_NumSpheresTouching.at(nSDs - 1);

    unsigned int* out_ptr = SD_SphereCompositeOffsets.data();
    unsigned int* in_ptr = SD_NumSpheresTouching.data();

    // copy data into the tmp array
    gpuErrchk(hipMemcpy(out_ptr, in_ptr, nSDs * sizeof(unsigned int), hipMemcpyDeviceToDevice));
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, in_ptr, out_ptr, nSDs);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
    // Allocate temporary storage
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, in_ptr, out_ptr, nSDs);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
    // total number of sphere entries to record
    unsigned int num_entries = out_ptr[nSDs - 1] + in_ptr[nSDs - 1];
    spheres_in_SD_composite.resize(num_entries, NULL_GRANULAR_ID);

    // make sure the DEs pointer is updated
    packSphereDataPointers(sphere_data);

    // printf("first run: num entries is %u, theoretical max is %u\n", num_entries, nSDs * MAX_COUNT_OF_SPHERES_PER_SD);

    // for (unsigned int i = 0; i < nSDs; i++) {
    //     printf("SD %d has offset %u, N %u \n", i, out_ptr[i], in_ptr[i]);
    // }

    // back up the offsets
    // TODO use a cached allocator, CUB provides one
    std::vector<unsigned int, cudallocator<unsigned int>> SD_SphereCompositeOffsets_bak;
    SD_SphereCompositeOffsets_bak.resize(SD_SphereCompositeOffsets.size());
    gpuErrchk(hipMemcpy(SD_SphereCompositeOffsets_bak.data(), SD_SphereCompositeOffsets.data(),
                         nSDs * sizeof(unsigned int), hipMemcpyDeviceToDevice));

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    sphereBroadphase<CUDA_THREADS_PER_BLOCK>
        <<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data, nSpheres, gran_params, num_entries);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    //
    // for (unsigned int i = 0; i < nSDs; i++) {
    //     printf("SD %d has offset %u, N %u \n", i, out_ptr[i], in_ptr[i]);
    // }
    //
    // for (unsigned int i = 0; i < num_entries; i++) {
    //     printf("entry %u is %u\n", i, spheres_in_SD_composite[i]);
    // }

    // restore the old offsets
    gpuErrchk(hipMemcpy(SD_SphereCompositeOffsets.data(), SD_SphereCompositeOffsets_bak.data(),
                         nSDs * sizeof(unsigned int), hipMemcpyDeviceToDevice));
    gpuErrchk(hipFree(d_temp_storage));
}

__host__ double ChSystemGranular_MonodisperseSMC::advance_simulation(float duration) {
    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    // Settling simulation loop.
    float duration_SU = duration / gran_params->TIME_UNIT;
    determineNewStepSize_SU();  // doesn't always change the timestep
    unsigned int nsteps = std::round(duration_SU / stepSize_SU);

    VERBOSE_PRINTF("advancing by %f at timestep %f, %u timesteps at approx user timestep %f\n", duration_SU,
                   stepSize_SU, nsteps, duration / nsteps);
    float time_elapsed_SU = 0;  // time elapsed in this advance call

    sphereDataStruct sphere_data;
    packSphereDataPointers(sphere_data);

    // Run the simulation, there are aggressive synchronizations because we want to have no race conditions
    for (; time_elapsed_SU < stepSize_SU * nsteps; time_elapsed_SU += stepSize_SU) {
        determineNewStepSize_SU();  // doesn't always change the timestep

        // Update the position and velocity of the BD, if relevant
        updateBDPosition(stepSize_SU);
        updateBCPositions();

        runSphereBroadphase();
        packSphereDataPointers(sphere_data);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        resetSphereAccelerations();
        resetBCForces();

        VERBOSE_PRINTF("Starting computeSphereForces!\n");

        if (gran_params->friction_mode == FRICTIONLESS) {
            // Compute sphere-sphere forces
            computeSphereForces_frictionless<<<nSDs, MAX_COUNT_OF_SPHERES_PER_SD>>>(
                sphere_data, gran_params, BC_type_list.data(), BC_params_list_SU.data(), BC_params_list_SU.size());
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        } else if (gran_params->friction_mode == SINGLE_STEP || gran_params->friction_mode == MULTI_STEP) {
            // figure out who is contacting
            determineContactPairs<<<nSDs, MAX_COUNT_OF_SPHERES_PER_SD>>>(sphere_data, gran_params);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());

            computeSphereContactForces<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(
                sphere_data, gran_params, BC_type_list.data(), BC_params_list_SU.data(), BC_params_list_SU.size(),
                nSpheres);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        }

        VERBOSE_PRINTF("Starting updatePositions!\n");
        updatePositions<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(stepSize_SU, sphere_data, nSpheres, gran_params);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        elapsedSimTime += stepSize_SU * gran_params->TIME_UNIT;  // Advance current time
    }

    return time_elapsed_SU * gran_params->TIME_UNIT;  // return elapsed UU time
}
}  // namespace granular
}  // namespace chrono
