#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2018 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Dan Negrut, Conlain Kelly, Nic Olsen
// =============================================================================
/*! \file */

#include <hip/hip_runtime.h>
#include "chrono_granular/ChGranularDefines.h"
#include "chrono_granular/physics/ChGranularCollision.cuh"
#include "chrono_granular/physics/ChGranularTriMesh.h"
#include "chrono_granular/utils/ChCudaMathUtils.cuh"
#include "chrono_granular/utils/ChGranularUtilities_CUDA.cuh"
#include "chrono_thirdparty/hipcub/hipcub.hpp"

// TODO should this go here?
// NOTE warpSize is a cuda environment value, but it is cc-dependent
#define warp_size 32

#define CUDA_THREADS 128

// These are the max X, Y, Z dimensions in the BD frame
#define MAX_X_POS_UNSIGNED (d_SD_Ldim_SU * d_box_L_SU)
#define MAX_Y_POS_UNSIGNED (d_SD_Ddim_SU * d_box_D_SU)
#define MAX_Z_POS_UNSIGNED (d_SD_Hdim_SU * d_box_H_SU)

#define Min(a, b) (a < b) ? a : b
#define Max(a, b) (a > b) ? a : b

#define Triangle_Soup chrono::granular::ChTriangleSoup

/// Takes in a triangle's position and finds out what SDs it touches
__device__ void triangle_figureOutTouchedSDs(unsigned int triangleID,
                                             const Triangle_Soup<int>& triangleSoup,
                                             unsigned int* touchedSDs) {
    unsigned int SD_count = 0;
    float3 vA, vB, vC;
    // Coalesced memory accesses; we have an int to float conversion here
    vA.x = triangleSoup.node1_X[triangleID];
    vA.y = triangleSoup.node1_Y[triangleID];
    vA.z = triangleSoup.node1_Z[triangleID];

    vB.x = triangleSoup.node2_X[triangleID];
    vB.y = triangleSoup.node2_Y[triangleID];
    vB.z = triangleSoup.node2_Z[triangleID];

    vC.x = triangleSoup.node3_X[triangleID];
    vC.y = triangleSoup.node3_Y[triangleID];
    vC.z = triangleSoup.node3_Z[triangleID];

    uint3 SDA = pointSDTriplet(vA.x, vA.y, vA.z);  // SD indices for point A
    uint3 SDB = pointSDTriplet(vB.x, vB.y, vB.z);  // SD indices for point B
    uint3 SDC = pointSDTriplet(vC.x, vC.y, vC.z);  // SD indices for point C

    uint3 L;  // Min SD index along each axis
    uint3 U;  // Max SD index along each axis

    L[0] = Min(SDA.x, Min(SDB.x, SDC.x));
    L[1] = Min(SDA.y, Min(SDB.y, SDC.y));
    L[2] = Min(SDA.z, Min(SDB.z, SDC.z));

    U[0] = Max(SDA.x, Max(SDB.x, SDC.x));
    U[1] = Max(SDA.y, Max(SDB.y, SDC.y));
    U[2] = Max(SDA.z, Max(SDB.z, SDC.z));

    // Case 1: All vetices are in the same SD
    if (L[0] == U[0] && L[1] == U[1] && L[2] == U[2]) {
        touchedSDs[SD_count++] = SDTripletID(L);
        return;
    }

    unsigned int n_axes_diff = 0;  // Count axes that have different SD bounds
    unsigned int axis_diff;

    for (unsigned int i = 0; i < 3; i++) {
        if (L[i] != U[i]) {
            axes_diff = i;  // If there is more than one, it won't be used anyway
            n_axes_diff++;
        }
    }

    // Case 2: Triangle lies in a Nx1x1, 1xNx1, or 1x1xN block of SDs
    if (n_axes_diff == 1) {
        uint3 SD_i = L;
        for (unsigned int i = L[axes_diff]; i <= U[axes_diff]; i++) {
            SD_i[axes_diff] = i;
            touchedSDs[SD_count++] = SDTripletID(SD_i);
        }
        return;
    }

    // Case 3: Triangle spans more than one dimension of nSD_spheres
    float SDcenter[3];
    float SDhalfSizes[3];
    uint3 SD_i = L;
    for (unsigned int i = L[0]; i <= U[0]; i++) {
        for (unsigned int j = L[1]; j <= U[1]; j++) {
            for (unsigned int k = L[2]; k <= U[2]; k++) {
                uint3 SD_i(i, j, k);
                SDhalfSizes[0] = d_SD_Ldim_SU;
                SDhalfSizes[1] = d_SD_Ddim_SU;
                SDhalfSizes[2] = d_SD_Hdim_SU;

                SDcenter[0] = d_BD_frame_X + (i * 2 + 1) * SDhalfSizes[0];
                SDcenter[1] = d_BD_frame_Y + (j * 2 + 1) * SDhalfSizes[1];
                SDcenter[2] = d_BD_frame_Z + (k * 2 + 1) * SDhalfSizes[2];

                if (check_TriangleBoxOverlap(SDcenter, SDhalfSizes, vA, vB, vC)) {
                    touchedSDs[SD_count++] = SDTripletID(SD_i);
                }
            }
        }
    }
}

/**
 * This kernel call prepares information that will be used in a subsequent kernel that performs the actual time
 * stepping.
 *
 * Nomenclature:
 *   - SD: subdomain.
 *   - BD: the big-domain, which is the union of all SDs
 *   - NULL_GRANULAR_ID: the equivalent of a non-sphere SD ID, or a non-sphere ID
 *
 * Template arguments:
 *   - CUB_THREADS: the number of threads used in this kernel, comes into play when invoking CUB block collectives
 *
 * Arguments:
 * SD_countsOfTrianglesTouching - array that for each SD indicates how many triangles touch this SD
 * triangles_in_SD_composite - big array that works in conjunction with SD_countsOfTrianglesTouching.
 *
 * Assumptions:
 *   - The size of the SD for the granular material and for the mesh is the same.
 *   - The SDs defining a CPB are a subset of the SDs spanning the terrain (granular material)
 *   - Each CPB has dimensions L x D x H.
 *   - The reference frame associated with the AABB that a CPB is:
 *       - The x-axis is along the length L of the box
 *       - The y-axis is along the width D of the box
 *       - The z-axis is along the height H of the box
 *       - The origin of the CPB is in a corner of the box
 *   - A mesh triangle cannot touch more than eight SDs
 *
 * Basic idea: use domain decomposition on the rectangular box and figure out how many SDs each triangle touches.
 * The subdomains are axis-aligned relative to the reference frame associated with the *box*. The origin of the box
 * is in the corner of the box. Each CPB is an AAB.
 *
 */
template <unsigned int CUB_THREADS>  //!< Number of threads engaged in block-collective CUB operations (multiple of 32)
__global__ void triangleSoupBroadPhase(
    Triangle_Soup<int>& d_triangleSoup,
    unsigned int*
        BUCKET_countsOfTrianglesTouching,  //!< Array that for each SD indicates how many triangles touch this SD
    unsigned int*
        triangles_in_BUCKET_composite  //!< Big array that works in conjunction with SD_countsOfTrianglesTouching.
                                       //!< "triangles_in_SD_composite" says which SD contains what triangles.
) {
    /// Set aside shared memory
    volatile __shared__ unsigned int offsetInComposite_TriangleInSD_Array[CUB_THREADS * MAX_SDs_TOUCHED_BY_TRIANGLE];
    volatile __shared__ bool shMem_head_flags[CUB_THREADS * MAX_SDs_TOUCHED_BY_TRIANGLE];

    typedef hipcub::BlockRadixSort<unsigned int, CUB_THREADS, MAX_SDs_TOUCHED_BY_TRIANGLE, unsigned int> BlockRadixSortOP;
    __shared__ typename BlockRadixSortOP::TempStorage temp_storage_sort;

    typedef hipcub::BlockDiscontinuity<unsigned int, CUB_THREADS> Block_Discontinuity;
    __shared__ typename Block_Discontinuity::TempStorage temp_storage_disc;

    unsigned int triangleIDs[MAX_SDs_TOUCHED_BY_TRIANGLE];
    unsigned int SDsTouched[MAX_SDs_TOUCHED_BY_TRIANGLE];

    // Figure out what triangleID this thread will handle. We work with a 1D block structure and a 1D grid structure
    unsigned int myTriangleID = threadIdx.x + blockIdx.x * blockDim.x;
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++) {
        /// start with a clean slate
        triangleIDs[i] = myTriangleID;
        SDsTouched[i] = NULL_GRANULAR_ID;
    }

    if (myTriangleID < d_triangleSoup.nTrianglesInSoup)
        triangle_figureOutTouchedSDs(myTriangleID, d_triangleSoup, SDsTouched);

    __syncthreads();

    // Truth be told, we are not interested in SDs touched, but rather buckets touched. This next step associates
    // SDs with "buckets". To save memory, since most SDs have no triangles, we "randomly" associate sevearl SDs
    // with a bucket. While the assignment of SDs to buckets is "random," the assignment scheme is deterministic:
    // for instance, SD 239 would always go to bucket 71.
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++)
        SDsTouched[i] = hashmapTagGenerator(SDsTouched[i]) % TRIANGLEBUCKET_COUNT;

    // Sort by the ID of the bucket touched
    BlockRadixSortOP(temp_storage_sort).Sort(SDsTouched, triangleIDs);
    __syncthreads();

    // Do a winningStreak search on whole block, might not have high utilization here
    bool head_flags[MAX_SDs_TOUCHED_BY_TRIANGLE];
    Block_Discontinuity(temp_storage_disc).FlagHeads(head_flags, SDsTouched, hipcub::Inequality());
    __syncthreads();

    // Write back to shared memory; eight-way bank conflicts here - to revisit later
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++) {
        shMem_head_flags[MAX_SDs_TOUCHED_BY_TRIANGLE * threadIdx.x + i] = head_flags[i];
    }

    // Seed offsetInComposite_TriangleInSD_Array with "no valid ID" so that we know later on what is legit;
    // No shmem bank coflicts here, good access...
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++) {
        offsetInComposite_TriangleInSD_Array[i * CUB_THREADS + threadIdx.x] = NULL_GRANULAR_ID_LONG;
    }

    __syncthreads();

    // Count how many times a Bucket shows up in conjunction with the collection of CUB_THREADS spheres. There
    // will be some thread divergence here.
    // Loop through each potential SD, after sorting, and see if it is the start of a head
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++) {
        // SD currently touched, could easily be inlined
        unsigned int touchedBucket = SDsTouched[i];
        if (touchedBucket != NULL_GRANULAR_ID && head_flags[i]) {
            // current index into shared datastructure of length 8*CUB_THREADS, could easily be inlined
            unsigned int idInShared = MAX_SDs_TOUCHED_BY_TRIANGLE * threadIdx.x + i;
            unsigned int winningStreak = 0;
            // This is the beginning of a sequence of SDs with a new ID
            do {
                winningStreak++;
                // Go until we run out of threads on the warp or until we find a new head
            } while (idInShared + winningStreak < MAX_SDs_TOUCHED_BY_TRIANGLE * CUB_THREADS &&
                     !(shMem_head_flags[idInShared + winningStreak]));

            // if (touchedSD >= d_box_L_SU * d_box_D_SU * d_box_H_SU) {
            //     printf("invalid SD index %u on thread %u\n", mySphereID, touchedSD);
            // }

            // Store start of new entries
            unsigned int offset = atomicAdd(BUCKET_countsOfTrianglesTouching + touchedBucket, winningStreak);

            // The value offset now gives a *relative* offset in the composite array; i.e., spheres_in_SD_composite.
            // Get the absolute offset
            offset += touchedBucket * MAX_COUNT_OF_Triangles_PER_SD;

            // Produce the offsets for this streak of spheres with identical SD ids
            for (unsigned int i = 0; i < winningStreak; i++)
                offsetInComposite_TriangleInSD_Array[idInShared + i] = offset++;
        }
    }

    __syncthreads();  // needed since we write to shared memory above; i.e., offsetInComposite_SphInSD_Array

    // Write out the data now; register with triangles_in_SD_composite each sphere that touches a certain ID
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++) {
        unsigned int offset = offsetInComposite_TriangleInSD_Array[MAX_SDs_TOUCHED_BY_TRIANGLE * threadIdx.x + i];
        if (offset != NULL_GRANULAR_ID_LONG) {
            triangles_in_BUCKET_composite[offset] = triangleIDs[i];
        }
    }
}

/**
This kernel call figures out forces on a sphere and carries out numerical integration to get the velocity updates of a
sphere.
N_CUDATHREADS - Number of threads in a CUDA block
MAX_NSPHERES_PER_SD - Max number of elements per SD. Shoudld be a power of two
MAX_TRIANGLES_PER_SD - Max number of elements per SD. Shoudld be a power of two
TRIANGLE_FAMILIES - The number of families that the triangles can belong to

Overview of implementation: One warp of threads will work on 32 triangles at a time to figure out the force that
they impress on a particular sphere. Note that each sphere enlists the services of one warp. If there are, say,
73 triangles touching this SD, it will take three trips to figure out the total force that the triangles will
impress upon the sphere that is active. If there are 256 threads in the block, then there will be 8 "active"
spheres since there are 8 warps in the block. Each thread in the block has enough registers to accummulate the
force felt by each "family", force that is the result of an interaction between a triangle and a sphere.
Say if sphere 232 touches a triangle that belongs to family 2, then a set of 6 generalized forces is going to
be produced to account for the interaction between the said triangle and sphere 232.
*/
#define Terrain chrono::granular::ChManyBodyStateWrapper

template <unsigned int N_CUDATHREADS,
          unsigned int MAX_NSPHERES_PER_SD,
          unsigned int MAX_TRIANGLES_PER_SD,
          unsigned int TRIANGLE_FAMILIES>
__global__ void interactionTerrain_TriangleSoup(
    Triangle_Soup<int>& d_triangleSoup,          //!< Contains information pertaining to triangle soup (in device mem.)
    Terrain& d_terrain,                          //!< Wrapper that stores terrain information available on the device
    unsigned int* SD_countsOfTrianglesTouching,  //!< Array that for each SD indicates how many triangles touch this SD
    unsigned int*
        triangles_in_SD_composite,  //!< Big array that works in conjunction with SD_countsOfTrianglesTouching.
                                    //!< "triangles_in_SD_composite" says which SD contains what triangles.
    unsigned int*
        SD_countsOfGrElemsTouching,         //!< Array that for each SD indicates how many grain elements touch this SD
    unsigned int* grElems_in_SD_composite)  //!< Big array that works in conjunction with SD_countsOfGrElemsTouching.
                                            //!< "grElems_in_SD_composite" says which SD contains what grElements.
{
    __shared__ unsigned int grElemID[MAX_NSPHERES_PER_SD];  //!< global ID of the grElements touching this SD
    __shared__ unsigned int triangID[MAX_NSPHERES_PER_SD];  //!< global ID of the triangles touching this SD

    __shared__ int sphX[MAX_NSPHERES_PER_SD];  //!< X coordinate of the grElement
    __shared__ int sphY[MAX_NSPHERES_PER_SD];  //!< Y coordinate of the grElement
    __shared__ int sphZ[MAX_NSPHERES_PER_SD];  //!< Z coordinate of the grElement

    __shared__ int node1_X[MAX_TRIANGLES_PER_SD];  //!< X coordinate of the 1st node of the triangle
    __shared__ int node1_Y[MAX_TRIANGLES_PER_SD];  //!< Y coordinate of the 1st node of the triangle
    __shared__ int node1_Z[MAX_TRIANGLES_PER_SD];  //!< Z coordinate of the 1st node of the triangle

    __shared__ int node2_X[MAX_TRIANGLES_PER_SD];  //!< X coordinate of the 2nd node of the triangle
    __shared__ int node2_Y[MAX_TRIANGLES_PER_SD];  //!< Y coordinate of the 2nd node of the triangle
    __shared__ int node2_Z[MAX_TRIANGLES_PER_SD];  //!< Z coordinate of the 2nd node of the triangle

    __shared__ int node3_X[MAX_TRIANGLES_PER_SD];  //!< X coordinate of the 3rd node of the triangle
    __shared__ int node3_Y[MAX_TRIANGLES_PER_SD];  //!< Y coordinate of the 3rd node of the triangle
    __shared__ int node3_Z[MAX_TRIANGLES_PER_SD];  //!< Z coordinate of the 3rd node of the triangle

    volatile __shared__ float tempShMem[6 * (N_CUDATHREADS / warp_size)];  // used to do a block-level reduce

    float forceActingOnSphere[3];  //!< 3 registers will hold the value of the force on the sphere
    float genForceActingOnMeshes[TRIANGLE_FAMILIES * 6];  //!< 6 components per family: 3 forces and 3 torques

    unsigned int thisSD = blockIdx.x;
    unsigned int nSD_triangles = SD_countsOfTrianglesTouching[thisSD];
    unsigned int nSD_spheres = SD_countsOfGrElemsTouching[thisSD];

    if (nSD_triangles == 0 || nSD_spheres == 0)
        return;

    // Populate the shared memory with terrain data
    unsigned int tripsToCoverSpheres = (nSD_spheres + blockDim.x - 1) / blockDim.x;
    unsigned int local_ID = threadIdx.x;
    for (unsigned int sphereTrip = 0; sphereTrip < tripsToCoverSpheres; sphereTrip++) {
        local_ID += sphereTrip * blockDim.x;
        if (local_ID < nSD_spheres) {
            unsigned int globalID = grElems_in_SD_composite[local_ID + thisSD * MAX_NSPHERES_PER_SD];
            grElemID[local_ID] = globalID;
            sphX[local_ID] = d_terrain.grElem_X[globalID];
            sphY[local_ID] = d_terrain.grElem_Y[globalID];
            sphZ[local_ID] = d_terrain.grElem_Z[globalID];
        }
    }
    // Populate the shared memory with mesh triangle data
    unsigned int tripsToCoverTriangles = (nSD_triangles + blockDim.x - 1) / blockDim.x;
    local_ID = threadIdx.x;
    for (unsigned int triangTrip = 0; triangTrip < tripsToCoverTriangles; triangTrip++) {
        local_ID += triangTrip * blockDim.x;
        if (local_ID < nSD_triangles) {
            unsigned int globalID = triangles_in_SD_composite[local_ID + thisSD * MAX_TRIANGLES_PER_SD];
            triangID[local_ID] = globalID;
            node1_X[local_ID] = d_triangleSoup.node1_X[globalID];
            node1_Y[local_ID] = d_triangleSoup.node1_Y[globalID];
            node1_Z[local_ID] = d_triangleSoup.node1_Z[globalID];

            node2_X[local_ID] = d_triangleSoup.node2_X[globalID];
            node2_Y[local_ID] = d_triangleSoup.node2_Y[globalID];
            node2_Z[local_ID] = d_triangleSoup.node2_Z[globalID];

            node3_X[local_ID] = d_triangleSoup.node3_X[globalID];
            node3_Y[local_ID] = d_triangleSoup.node3_Y[globalID];
            node3_Z[local_ID] = d_triangleSoup.node3_Z[globalID];
        }
    }

    __syncthreads();  // this call ensures data is in its place in shared memory

    /// Zero out the force and torque at the onset of the computation
    for (local_ID = 0; local_ID < TRIANGLE_FAMILIES; local_ID++) {
        unsigned int dummyOffset = 6 * local_ID;
        /// forces acting on the triangle, in global reference frame
        genForceActingOnMeshes[dummyOffset++] = 0.f;
        genForceActingOnMeshes[dummyOffset++] = 0.f;
        genForceActingOnMeshes[dummyOffset++] = 0.f;
        /// torques with respect to global reference frame, expressed in global reference frame
        genForceActingOnMeshes[dummyOffset++] = 0.f;
        genForceActingOnMeshes[dummyOffset++] = 0.f;
        genForceActingOnMeshes[dummyOffset++] = 0.f;
    }

    // Each sphere has one warp of threads dedicated to identifying all triangles that this sphere
    // touches. Upon a contact event, we'll compute the normal force on the sphere; and, the force and torque
    // impressed upon the triangle

    unsigned int nSpheresProcessedAtOneTime =
        blockDim.x / warp_size;  /// One warp allocated to slave serving one sphere
    tripsToCoverSpheres = (nSD_spheres + nSpheresProcessedAtOneTime - 1) / nSpheresProcessedAtOneTime;
    tripsToCoverTriangles = (nSD_triangles + warp_size - 1) / warp_size;

    unsigned sphere_Local_ID = threadIdx.x / warp_size;
    for (unsigned int sphereTrip = 0; sphereTrip < tripsToCoverSpheres; sphereTrip++) {
        /// before starting dealing with a sphere, zero out the forces acting on it; all threads in the block are
        /// doing this
        forceActingOnSphere[0] = 0.f;
        forceActingOnSphere[1] = 0.f;
        forceActingOnSphere[2] = 0.f;
        sphere_Local_ID += sphereTrip * nSpheresProcessedAtOneTime;
        if (sphere_Local_ID < nSD_spheres) {
            /// Figure out which triangles this sphere collides with; each thread in a warp slaving for this sphere
            /// looks at one triangle at a time. The collection of threads in the warp sweeps through all the
            /// triangles that touch this SD. NOTE: to avoid double-counting, a sphere-triangle collision event is
            /// counted only if the collision point is in this SD.
            unsigned int targetTriangle = (threadIdx.x & (warp_size - 1));  // computes modulo 32 of the thread index
            for (unsigned int triangTrip = 0; triangTrip < tripsToCoverTriangles; triangTrip++) {
                targetTriangle += triangTrip * warp_size;
                if (targetTriangle < nSD_triangles) {
                    /// we have a valid sphere and a valid triganle; check if in contact
                    double3 norm;
                    double depth;
                    double3 pt1;
                    double3 pt2;
                    double eff_radius;
                    double3 A = make_double3(node1_X[targetTriangle], node1_Y[targetTriangle], node1_Z[targetTriangle]);
                    double3 B = make_double3(node2_X[targetTriangle], node2_Y[targetTriangle], node2_Z[targetTriangle]);
                    double3 C = make_double3(node3_X[targetTriangle], node3_Y[targetTriangle], node3_Z[targetTriangle]);
                    double3 sphCntr = make_double3(sphX[sphere_Local_ID], sphY[sphere_Local_ID], sphZ[sphere_Local_ID]);
                    face_sphere_cd(A, B, C, sphCntr, d_terrain.sphereRadius, norm, depth, pt1, pt2, eff_radius);

                    /// Use the CD information to compute the force on the grElement

                    /// Use the CD information to compute the force and torque on the triangle
                }
            }
            /// down to the point where we need to collect the forces from all the threads in the wrap; this is a
            /// warp reduce operation. The resultant force acting on this grElement is stored in the first lane of
            /// the warp. NOTE: In this warp-level operations participate only the warps that are slaving for a
            /// sphere; i.e., some warps see no action
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[0] += __shfl_down_sync(0xffffffff, forceActingOnSphere[0], offset);
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[1] += __shfl_down_sync(0xffffffff, forceActingOnSphere[1], offset);
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[2] += __shfl_down_sync(0xffffffff, forceActingOnSphere[2], offset);

            /// done with the computation of all the contacts that the triangles impress on this sphere. Update the
            /// position of the sphere based on this force
        }
    }
    /// Done computing the forces acting on the triangles in this SD. A block reduce is carried out next. Start by
    /// doing a reduce at the warp level.
    for (local_ID = 0; local_ID < TRIANGLE_FAMILIES; local_ID++) {
        /// six generalized forces acting on the triangle, expressed in the global reference frame
        unsigned int dummyIndx = 6 * local_ID;
        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
        dummyIndx++;

        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
        dummyIndx++;

        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
        dummyIndx++;

        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
        dummyIndx++;

        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
        dummyIndx++;

        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
    }

    __syncthreads();

    /// Lane zero in each warp holds the result of a warp-level reduce operation. Sum up these "Lane zero" values in
    /// the final result, which is block-level
    bool threadIsLaneZeroInWarp = ((threadIdx.x & (warp_size - 1)) == 0);
    for (local_ID = 0; local_ID < TRIANGLE_FAMILIES; local_ID++) {
        unsigned int offsetGenForceArray = 6 * local_ID;
        /// Place in ShMem forces/torques (expressed in global reference frame) acting on this family of triangles
        if (threadIsLaneZeroInWarp) {
            unsigned int offsetShMem = 6 * (threadIdx.x / warp_size);
            tempShMem[offsetShMem++] = genForceActingOnMeshes[offsetGenForceArray++];
            tempShMem[offsetShMem++] = genForceActingOnMeshes[offsetGenForceArray++];
            tempShMem[offsetShMem++] = genForceActingOnMeshes[offsetGenForceArray++];

            tempShMem[offsetShMem++] = genForceActingOnMeshes[offsetGenForceArray++];
            tempShMem[offsetShMem++] = genForceActingOnMeshes[offsetGenForceArray++];
            tempShMem[offsetShMem] = genForceActingOnMeshes[offsetGenForceArray];
        }
        __syncthreads();

        /// Going to trash the values in "forceActingOnSphere", not needed anymore. Reuse the registers, which will
        /// now store the vaule of the triangle force and torque...
        if (threadIdx.x < warp_size) {
            /// only first thread in block participates in this reduce operation.
            /// NOTE: an implicit assumption is made here - warp_size is larger than or equal to N_CUDATHREADS /
            /// warp_size. This is true today as N_CUDATHREADS cannot be larger than 1024 and warp_size is 32.

            /// Work on forces first. Place data from ShMem into registers associated w/ first warp
            unsigned int offsetShMem = 6 * threadIdx.x;
            if (threadIdx.x < (N_CUDATHREADS / warp_size)) {
                forceActingOnSphere[0] = tempShMem[offsetShMem++];
                forceActingOnSphere[1] = tempShMem[offsetShMem++];
                forceActingOnSphere[2] = tempShMem[offsetShMem++];
            } else {
                /// this is hit only by a subset of threads from first warp of the block
                forceActingOnSphere[0] = 0.f;
                forceActingOnSphere[1] = 0.f;
                forceActingOnSphere[2] = 0.f;
            }

            offsetGenForceArray = 6 * local_ID;
            // X component of the force on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[0] += __shfl_down_sync(0xffffffff, forceActingOnSphere[0], offset);
            genForceActingOnMeshes[offsetGenForceArray++] = forceActingOnSphere[0];

            // Y component of the force on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[1] += __shfl_down_sync(0xffffffff, forceActingOnSphere[1], offset);
            genForceActingOnMeshes[offsetGenForceArray++] = forceActingOnSphere[1];

            // Z component of the force on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[2] += __shfl_down_sync(0xffffffff, forceActingOnSphere[2], offset);
            genForceActingOnMeshes[offsetGenForceArray++] = forceActingOnSphere[2];

            /// Finally, work on torques
            if (threadIdx.x < (N_CUDATHREADS / warp_size)) {
                forceActingOnSphere[0] = tempShMem[offsetShMem++];
                forceActingOnSphere[1] = tempShMem[offsetShMem++];
                forceActingOnSphere[2] = tempShMem[offsetShMem];
            } else {
                /// this is hit only by a subset of threads from first warp of the block
                forceActingOnSphere[0] = 0.f;
                forceActingOnSphere[1] = 0.f;
                forceActingOnSphere[2] = 0.f;
            }

            // X component of the torque on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[0] += __shfl_down_sync(0xffffffff, forceActingOnSphere[0], offset);
            genForceActingOnMeshes[offsetGenForceArray++] = forceActingOnSphere[0];

            // Y component of the torque on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[1] += __shfl_down_sync(0xffffffff, forceActingOnSphere[1], offset);
            genForceActingOnMeshes[offsetGenForceArray++] = forceActingOnSphere[1];

            // Z component of the torque on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[2] += __shfl_down_sync(0xffffffff, forceActingOnSphere[2], offset);
            genForceActingOnMeshes[offsetGenForceArray] = forceActingOnSphere[2];
        }  /// this is the end of the "for each mesh" loop

        /// At this point, the first thread of the block has in genForceActingOnMeshes[6*TRIANGLE_FAMILIES] the
        /// forces and torques acting on each mesh family. Bcast the force values to all threads in the warp.
        /// To this end, synchronize all threads in warp and get "value" from lane 0
        for (local_ID = 0; local_ID < 6 * TRIANGLE_FAMILIES; local_ID++)
            genForceActingOnMeshes[local_ID] = __shfl_sync(0xffffffff, genForceActingOnMeshes[local_ID], 0);

        /// At this point, all threads in the first warp have the generalized forces acting on all meshes. Do an
        /// atomic add to compund the value of the generalized forces acting on the meshes that come in contact with
        /// the granular material.
        unsigned int nTrips = (6 * TRIANGLE_FAMILIES) / warp_size;
        for (local_ID = 0; local_ID < nTrips + 1; local_ID++) {
            unsigned int offset = threadIdx.x + local_ID * (6 * TRIANGLE_FAMILIES);
            if (offset < 6 * TRIANGLE_FAMILIES)
                atomicAdd(d_triangleSoup.generalizedForcesPerFamily + offset, genForceActingOnMeshes[offset]);
        }
    }
}

/// Copy const triangle data to device
void chrono::granular::ChSystemGranularMonodisperse_SMC_Frictionless_trimesh::copy_triangle_data_to_device() {
    // Handle what's specific to the case when the mesh is present
    // gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_Kn_s2m_SU), &K_n_s2m_SU, sizeof(d_Kn_s2m_SU)));
}

__host__ void chrono::granular::ChSystemGranularMonodisperse_SMC_Frictionless_trimesh::run_simulation(float tEnd) {
    NOT_IMPLEMENTED_YET;
}

__host__ void chrono::granular::ChSystemGranularMonodisperse_SMC_Frictionless_trimesh::advance_simulation(
    float duration) {
    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nDEs + CUDA_THREADS - 1) / CUDA_THREADS;

    // Settling simulation loop.
    unsigned int stepSize_SU = 5;
    unsigned int duration_SU = std::ceil(duration / (TIME_UNIT * PSI_h));
    unsigned int nsteps = (1.0 * duration_SU) / stepSize_SU;

    printf("advancing by %u at timestep %u, %u timesteps at approx user timestep %f\n", duration_SU, stepSize_SU,
           nsteps, duration / nsteps);
    printf("z grav term with timestep %u is %f\n", stepSize_SU, stepSize_SU * stepSize_SU * gravity_Z_SU);

    VERBOSE_PRINTF("Starting Main Simulation loop!\n");
    // Run the simulation, there are aggressive synchronizations because we want to have no race conditions
    for (unsigned int crntTime_SU = 0; crntTime_SU < stepSize_SU * nsteps; crntTime_SU += stepSize_SU) {
        /// DO STEP LOOP HERE

        // // reset forces to zero, note that vel update ~ force for forward euler
        // gpuErrchk(hipMemset(pos_X_dt_update.data(), 0, nDEs * sizeof(float)));
        // gpuErrchk(hipMemset(pos_Y_dt_update.data(), 0, nDEs * sizeof(float)));
        // gpuErrchk(hipMemset(pos_Z_dt_update.data(), 0, nDEs * sizeof(float)));
        //
        // VERBOSE_PRINTF("Starting computeVelocityUpdates!\n");
        //
        // // Compute forces and crank into vel updates, we have 2 kernels to avoid a race condition
        // computeVelocityUpdates<MAX_COUNT_OF_DEs_PER_SD><<<nSDs, MAX_COUNT_OF_DEs_PER_SD>>>(
        //     stepSize_SU, pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt_update.data(),
        //     pos_Y_dt_update.data(), pos_Z_dt_update.data(), SD_NumOf_DEs_Touching.data(),
        //     DEs_in_SD_composite.data(), pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data());
        // gpuErrchk(hipPeekAtLastError());
        // gpuErrchk(hipDeviceSynchronize());
        //
        // VERBOSE_PRINTF("Starting applyVelocityUpdates!\n");
        // // Apply the updates we just made
        // applyVelocityUpdates<MAX_COUNT_OF_DEs_PER_SD><<<nSDs, MAX_COUNT_OF_DEs_PER_SD>>>(
        //     stepSize_SU, pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt_update.data(),
        //     pos_Y_dt_update.data(), pos_Z_dt_update.data(), SD_NumOf_DEs_Touching.data(),
        //     DEs_in_SD_composite.data(), pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data());
        //
        // gpuErrchk(hipPeekAtLastError());
        // gpuErrchk(hipDeviceSynchronize());
        // VERBOSE_PRINTF("Resetting broadphase info!\n");
        //
        // // Reset broadphase information
        // resetBroadphaseInformation();
        //
        // VERBOSE_PRINTF("Starting updatePositions!\n");
        // updatePositions<CUDA_THREADS><<<nBlocks, CUDA_THREADS>>>(
        //     stepSize_SU, pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt.data(), pos_Y_dt.data(),
        //     pos_Z_dt.data(), SD_NumOf_DEs_Touching.data(), DEs_in_SD_composite.data(), nDEs);
        //
        // gpuErrchk(hipPeekAtLastError());
        // gpuErrchk(hipDeviceSynchronize());
    }
    printf("SU radius is %u\n", sphereRadius_SU);
    // Don't write but print verbosely

    return;
}

void chrono::granular::ChSystemGranularMonodisperse_SMC_Frictionless_trimesh::cleanupSoup_DEVICE() {
    hipFree(meshSoup_DEVICE.triangleFamily_ID);

    hipFree(meshSoup_DEVICE.node1_X);
    hipFree(meshSoup_DEVICE.node1_Y);
    hipFree(meshSoup_DEVICE.node1_Z);

    hipFree(meshSoup_DEVICE.node2_X);
    hipFree(meshSoup_DEVICE.node2_Y);
    hipFree(meshSoup_DEVICE.node2_Z);

    hipFree(meshSoup_DEVICE.node3_X);
    hipFree(meshSoup_DEVICE.node3_Y);
    hipFree(meshSoup_DEVICE.node3_Z);

    hipFree(meshSoup_DEVICE.node1_XDOT);
    hipFree(meshSoup_DEVICE.node1_YDOT);
    hipFree(meshSoup_DEVICE.node1_ZDOT);

    hipFree(meshSoup_DEVICE.node2_XDOT);
    hipFree(meshSoup_DEVICE.node2_YDOT);
    hipFree(meshSoup_DEVICE.node2_ZDOT);

    hipFree(meshSoup_DEVICE.node3_XDOT);
    hipFree(meshSoup_DEVICE.node3_YDOT);
    hipFree(meshSoup_DEVICE.node3_ZDOT);

    hipFree(meshSoup_DEVICE.generalizedForcesPerFamily);
}

void chrono::granular::ChSystemGranularMonodisperse_SMC_Frictionless_trimesh::setupSoup_DEVICE(
    unsigned int nTriangles) {
    /// Allocate the DEVICE mesh soup
    meshSoup_DEVICE.nTrianglesInSoup = nTriangles;  // TODO this is not on device?

    gpuErrchk(hipMalloc(&meshSoup_DEVICE.triangleFamily_ID, nTriangles * sizeof(unsigned int)));

    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node1_X, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node1_Y, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node1_Z, nTriangles * sizeof(float)));

    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node2_X, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node2_Y, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node2_Z, nTriangles * sizeof(float)));

    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node3_X, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node3_Y, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node3_Z, nTriangles * sizeof(float)));

    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node1_XDOT, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node1_YDOT, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node1_ZDOT, nTriangles * sizeof(float)));

    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node2_XDOT, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node2_YDOT, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node2_ZDOT, nTriangles * sizeof(float)));

    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node3_XDOT, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node3_YDOT, nTriangles * sizeof(float)));
    gpuErrchk(hipMalloc(&meshSoup_DEVICE.node3_ZDOT, nTriangles * sizeof(float)));
}
