#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2018 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Dan Negrut, Conlain Kelly, Nic Olsen
// =============================================================================
/*! \file */
// These two must be included first
#include "chrono_granular/physics/ChGranularGPU_SMC.cuh"
#include "chrono_granular/physics/ChGranularTriMesh.h"
// these define things that mess with cub
#include "chrono_granular/physics/ChGranularCollision.cuh"
#include "chrono_granular/physics/ChGranularBoxTriangle.cuh"
#include "chrono_granular/utils/ChCudaMathUtils.cuh"

// TODO should this go here?
// NOTE warpSize is a cuda environment value, but it is cc-dependent
#if __CUDA_ARCH__ <= 600
// all devices of compute capability <= 6.0
static const int warp_size = 32;
#else
static const int warp_size = warpSize;
#endif

#define NUM_TRIANGLE_FAMILIES 4

#define Triangle_Soup chrono::granular::ChTriangleSoup

typedef const chrono::granular::ChSystemGranularMonodisperse_SMC_Frictionless_trimesh::GranParamsHolder_trimesh*
    MeshParamsPtr;

/// Takes in a triangle's position and finds out what SDs it touches
__device__ void triangle_figureOutTouchedSDs(unsigned int triangleID,
                                             const Triangle_Soup<float>* triangleSoup,
                                             unsigned int* touchedSDs,
                                             ParamsPtr gran_params) {
    unsigned int SD_count = 0;
    float3 vA, vB, vC;
    // Coalesced memory accesses; we have an int to float conversion here
    vA.x = triangleSoup->node1_X[triangleID];
    vA.y = triangleSoup->node1_Y[triangleID];
    vA.z = triangleSoup->node1_Z[triangleID];

    vB.x = triangleSoup->node2_X[triangleID];
    vB.y = triangleSoup->node2_Y[triangleID];
    vB.z = triangleSoup->node2_Z[triangleID];

    vC.x = triangleSoup->node3_X[triangleID];
    vC.y = triangleSoup->node3_Y[triangleID];
    vC.z = triangleSoup->node3_Z[triangleID];

    uint3 SDA = pointSDTriplet(vA.x, vA.y, vA.z, gran_params);  // SD indices for point A
    uint3 SDB = pointSDTriplet(vB.x, vB.y, vB.z, gran_params);  // SD indices for point B
    uint3 SDC = pointSDTriplet(vC.x, vC.y, vC.z, gran_params);  // SD indices for point C

    unsigned int L[3];  // Min SD index along each axis
    unsigned int U[3];  // Max SD index along each axis

    L[0] = MIN(SDA.x, MIN(SDB.x, SDC.x));
    L[1] = MIN(SDA.y, MIN(SDB.y, SDC.y));
    L[2] = MIN(SDA.z, MIN(SDB.z, SDC.z));

    U[0] = MAX(SDA.x, MAX(SDB.x, SDC.x));
    U[1] = MAX(SDA.y, MAX(SDB.y, SDC.y));
    U[2] = MAX(SDA.z, MAX(SDB.z, SDC.z));

    // Case 1: All vetices are in the same SD
    if (L[0] == U[0] && L[1] == U[1] && L[2] == U[2]) {
        touchedSDs[SD_count++] = SDTripletID(L, gran_params);
        return;
    }

    unsigned int n_axes_diff = 0;  // Count axes that have different SD bounds
    unsigned int axes_diff;

    for (unsigned int i = 0; i < 3; i++) {
        if (L[i] != U[i]) {
            axes_diff = i;  // If there is more than one, it won't be used anyway
            n_axes_diff++;
        }
    }

    // Case 2: Triangle lies in a Nx1x1, 1xNx1, or 1x1xN block of SDs
    if (n_axes_diff == 1) {
        unsigned int SD_i[3] = {L[0], L[1], L[2]};
        for (unsigned int i = L[axes_diff]; i <= U[axes_diff]; i++) {
            SD_i[axes_diff] = i;
            touchedSDs[SD_count++] = SDTripletID(SD_i, gran_params);
        }
        return;
    }

    // Case 3: Triangle spans more than one dimension of nSD_spheres
    float SDcenter[3];
    float SDhalfSizes[3];
    for (unsigned int i = L[0]; i <= U[0]; i++) {
        for (unsigned int j = L[1]; j <= U[1]; j++) {
            for (unsigned int k = L[2]; k <= U[2]; k++) {
                SDhalfSizes[0] = gran_params->d_SD_Ldim_SU;
                SDhalfSizes[1] = gran_params->d_SD_Ddim_SU;
                SDhalfSizes[2] = gran_params->d_SD_Hdim_SU;

                SDcenter[0] = gran_params->d_BD_frame_X + (i * 2 + 1) * SDhalfSizes[0];
                SDcenter[1] = gran_params->d_BD_frame_Y + (j * 2 + 1) * SDhalfSizes[1];
                SDcenter[2] = gran_params->d_BD_frame_Z + (k * 2 + 1) * SDhalfSizes[2];

                if (check_TriangleBoxOverlap(SDcenter, SDhalfSizes, vA, vB, vC)) {
                    touchedSDs[SD_count++] = SDTripletID(i, j, k, gran_params);
                }
            }
        }
    }
}

/**
 * This kernel carries out broadphase for the triangle soup
 *
 * Nomenclature:
 *   - SD: subdomain.
 *   - BD: the big-domain, which is the union of all SDs
 *   - NULL_GRANULAR_ID: the equivalent of a non-sphere SD ID, or a non-sphere ID
 *
 * Template arguments:
 *   - CUB_THREADS: the number of threads used in this kernel, comes into play when invoking CUB block collectives
 *
 * Arguments:
 * SD_countsOfTrianglesTouching - array that for each SD indicates how many triangles touch this SD
 * triangles_in_SD_composite - big array that works in conjunction with SD_countsOfTrianglesTouching.
 *
 * Assumptions:
 *   - The size of the SD for the granular material and for the mesh is the same.
 *   - A mesh triangle cannot touch more than MAX_SDs_TOUCHED_BY_TRIANGLE SDs
 *
 * Basic idea: use domain decomposition on the rectangular box and figure out how the buckets that each triangle
 * touches. The subdomains are axis-aligned relative to the reference frame associated with the *box*. The origin of the
 * box is in the corner of the box. Each CPB is an AAB.
 *
 */
template <unsigned int CUB_THREADS>  //!< Number of threads engaged in block-collective CUB operations (multiple of 32)
__global__ void triangleSoupBroadPhase(
    Triangle_Soup<float>* d_triangleSoup,
    unsigned int*
        BUCKET_countsOfTrianglesTouching,  //!< Array that for each SD indicates how many triangles touch this SD
    unsigned int*
        triangles_in_BUCKET_composite,  //!< Big array that works in conjunction with SD_countsOfTrianglesTouching.
                                        //!< "triangles_in_SD_composite" says which SD contains what triangles.
    unsigned int* SD_countsOfTrianglesTouching,  //!< If SD 629 has any triangle touching it, then
                                                 //!< SD_countsOfTrianglesTouching[629]>0.
    ParamsPtr gran_params,
    MeshParamsPtr mesh_params) {
    /// Set aside shared memory
    volatile __shared__ unsigned int offsetInComposite_TriangleInBKT_Array[CUB_THREADS * MAX_SDs_TOUCHED_BY_TRIANGLE];
    volatile __shared__ bool shMem_head_flags[CUB_THREADS * MAX_SDs_TOUCHED_BY_TRIANGLE];

    typedef hipcub::BlockRadixSort<unsigned int, CUB_THREADS, MAX_SDs_TOUCHED_BY_TRIANGLE, unsigned int> BlockRadixSortOP;
    __shared__ typename BlockRadixSortOP::TempStorage temp_storage_sort;

    typedef hipcub::BlockDiscontinuity<unsigned int, CUB_THREADS> Block_Discontinuity;
    __shared__ typename Block_Discontinuity::TempStorage temp_storage_disc;

    unsigned int triangleIDs[MAX_SDs_TOUCHED_BY_TRIANGLE];
    unsigned int SDsTouched[MAX_SDs_TOUCHED_BY_TRIANGLE];
    unsigned int BKTsTouched[MAX_SDs_TOUCHED_BY_TRIANGLE];

    // Figure out what triangleID this thread will handle. We work with a 1D block structure and a 1D grid structure
    unsigned int myTriangleID = threadIdx.x + blockIdx.x * blockDim.x;
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++) {
        /// start with a clean slate
        triangleIDs[i] = myTriangleID;
        SDsTouched[i] = NULL_GRANULAR_ID;
        BKTsTouched[i] = NULL_GRANULAR_ID;
    }

    if (myTriangleID < d_triangleSoup->nTrianglesInSoup) {
        triangle_figureOutTouchedSDs(myTriangleID, d_triangleSoup, SDsTouched, gran_params);
    }

    __syncthreads();

    // Truth be told, we are not interested in SDs touched, but rather buckets touched. This next step associates SDs
    // with "buckets". To save memory, since most SDs have no triangles, we "randomly" associate several SDs with a
    // bucket. While the assignment of SDs to buckets is "random," the assignment scheme is deterministic: for
    // instance, SD 239 would always go to bucket 71.
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++)
        if (SDsTouched[i] != NULL_GRANULAR_ID) {
            BKTsTouched[i] = hashmapBKTid(SDsTouched[i]) % TRIANGLEBUCKET_COUNT;
        }

    // Earmark SDs that are touched by at least one triangle. This step is needed since when computing the
    // mesh-GrMat interaction we only want to do narrow phase on an SD that actually is touched by triangles. Keep
    // in mind thta several SDs deposit their triangles in the same bucket. As such, later one during narrow phase/force
    // computation, if an SD looks for a bucket and sees triangles in there, if we know that this SD is touching zero
    // triangles then that SD is not going to do narrow phase on the triangles in that bucket since these triangles
    // actually are associated with other SDs that happen to deposit their triangles in this same bucket.
    // NOTE why are we sorting this? also we can't use this storage like that, it's for a key-value sort
    BlockRadixSortOP(temp_storage_sort).Sort(SDsTouched, triangleIDs);
    __syncthreads();

    // Do a winningStreak search on whole block, might not have high utilization here
    bool head_flags[MAX_SDs_TOUCHED_BY_TRIANGLE];
    Block_Discontinuity(temp_storage_disc).FlagHeads(head_flags, SDsTouched, hipcub::Inequality());
    __syncthreads();

    // If a thread is associated with a legit discontinuity; i.e., not one associated with NULL_GRANULAR_ID, it should
    // flag that SD as being touched by a triangle
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++) {
        if (head_flags[i] && (SDsTouched[i] != NULL_GRANULAR_ID))
            atomicAdd(SD_countsOfTrianglesTouching, 1);
    }

    // Back at working with buckets. For all purposes, the role that SDs play in this kernel is over.
    BlockRadixSortOP(temp_storage_sort).Sort(BKTsTouched, triangleIDs);
    __syncthreads();

    // Do a winningStreak search on whole block, might not have high utilization here
    Block_Discontinuity(temp_storage_disc).FlagHeads(head_flags, BKTsTouched, hipcub::Inequality());
    __syncthreads();

    // Write back to shared memory; eight-way bank conflicts here - to revisit later
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++) {
        shMem_head_flags[MAX_SDs_TOUCHED_BY_TRIANGLE * threadIdx.x + i] = head_flags[i];
    }

    // Seed offsetInComposite_TriangleInSD_Array with "no valid ID" so that we know later on what is legit;
    // No shmem bank coflicts here, good access...
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++) {
        offsetInComposite_TriangleInBKT_Array[i * CUB_THREADS + threadIdx.x] = NULL_GRANULAR_ID_LONG;
    }

    __syncthreads();

    // Count how many times a Bucket shows up in conjunction with the collection of CUB_THREADS triangles. There
    // will be some thread divergence here.
    // Loop through each potential BKT, after sorting, and see if it is the start of a head
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++) {
        // SD currently touched, could easily be inlined
        unsigned int touchedBucket = BKTsTouched[i];
        if (head_flags[i] && (touchedBucket != NULL_GRANULAR_ID)) {
            // current index into shared datastructure of length 8*CUB_THREADS, could easily be inlined
            unsigned int idInShared = MAX_SDs_TOUCHED_BY_TRIANGLE * threadIdx.x + i;
            unsigned int winningStreak = 0;
            // This is the beginning of a sequence of BKTs with a new ID
            do {
                winningStreak++;
                // Go until we run out of threads on the warp or until we find a new head
            } while (idInShared + winningStreak < MAX_SDs_TOUCHED_BY_TRIANGLE * CUB_THREADS &&
                     !(shMem_head_flags[idInShared + winningStreak]));

            // if (touchedSD >= d_box_L_SU * d_box_D_SU * d_box_H_SU) {
            //     printf("invalid SD index %u on thread %u\n", mySphereID, touchedSD);
            // }

            // Store start of new entries
            unsigned int offset = atomicAdd(BUCKET_countsOfTrianglesTouching + touchedBucket, winningStreak);

            // The value offset now gives a *relative* offset in the composite array.
            // Get the absolute offset
            offset += touchedBucket * MAX_TRIANGLE_COUNT_PER_BUCKET;

            // Produce the offsets for this streak of triangles with identical BKT ids
            for (unsigned int i = 0; i < winningStreak; i++)
                offsetInComposite_TriangleInBKT_Array[idInShared + i] = offset++;
        }
    }

    __syncthreads();  // needed since we write to shared memory above; i.e., offsetInComposite_SphInSD_Array

    // Write out the data now; register with triangles_in_SD_composite each sphere that touches a certain ID
    for (unsigned int i = 0; i < MAX_SDs_TOUCHED_BY_TRIANGLE; i++) {
        unsigned int offset = offsetInComposite_TriangleInBKT_Array[MAX_SDs_TOUCHED_BY_TRIANGLE * threadIdx.x + i];
        if (offset != NULL_GRANULAR_ID_LONG) {
            triangles_in_BUCKET_composite[offset] = triangleIDs[i];
        }
    }
}

/**
This kernel call figures out forces on a sphere and carries out numerical integration to get the velocity updates of a
sphere.
N_CUDATHREADS - Number of threads in a CUDA block
MAX_NSPHERES_PER_SD - Max number of elements per SD. Shoudld be a power of two
TRIANGLE_FAMILIES - The number of families that the triangles can belong to

Overview of implementation: One warp of threads will work on 32 triangles at a time to figure out the force that
they impress on a particular sphere. Note that each sphere enlists the services of one warp. If there are, say,
73 triangles touching this SD, it will take three trips to figure out the total force that the triangles will
impress upon the sphere that is active. If there are 256 threads in the block, then there will be 8 "active"
spheres since there are 8 warps in the block. Each thread in the block has enough registers to accummulate the
force felt by each "family", force that is the result of an interaction between a triangle and a sphere.
Say if sphere 232 touches a triangle that belongs to family 2, then a set of 6 generalized forces is going to
be produced to account for the interaction between the said triangle and sphere 232.
*/

// TODO fix this
#define TRIANGLE_FAMILIES 4

template <unsigned int N_CUDATHREADS>
__global__ void interactionTerrain_TriangleSoup(
    Triangle_Soup<float>* d_triangleSoup,  //!< Contains information pertaining to triangle soup (in device mem.)
    int* d_sphere_pos_X,
    int* d_sphere_pos_Y,
    int* d_sphere_pos_Z,
    float* d_sphere_pos_X_dt,
    float* d_sphere_pos_Y_dt,
    float* d_sphere_pos_Z_dt,
    unsigned int* BKT_countsOfTrianglesTouching,  //!< Array that for each SD indicates how many triangles touch this SD
    unsigned int*
        triangles_in_BKT_composite,  //!< Big array that works in conjunction with SD_countsOfTrianglesTouching.
                                     //!< "triangles_in_SD_composite" says which SD contains what triangles.
    unsigned int*
        SD_countsOfGrElemsTouching,         //!< Array that for each SD indicates how many grain elements touch this SD
    unsigned int* grElems_in_SD_composite,  //!< Big array that works in conjunction with SD_countsOfGrElemsTouching.
                                            //!< "grElems_in_SD_composite" says which SD contains what grElements.
    unsigned int* SD_countsOfTrianglesTouching,  //!< The length of this array is equal to number of SDs. If SD 423 is
                                                 //!< touched by any triangle, then SD_countsOfTrianglesTouching[423]>0.

    ParamsPtr gran_params,
    MeshParamsPtr mesh_params) {
    __shared__ unsigned int grElemID[MAX_COUNT_OF_DEs_PER_SD];        //!< global ID of the grElements touching this SD
    __shared__ unsigned int triangID[MAX_TRIANGLE_COUNT_PER_BUCKET];  //!< global ID of the triangles touching this SD

    __shared__ int sphX[MAX_COUNT_OF_DEs_PER_SD];  //!< X coordinate of the grElement
    __shared__ int sphY[MAX_COUNT_OF_DEs_PER_SD];  //!< Y coordinate of the grElement
    __shared__ int sphZ[MAX_COUNT_OF_DEs_PER_SD];  //!< Z coordinate of the grElement

    __shared__ int node1_X[MAX_TRIANGLE_COUNT_PER_BUCKET];  //!< X coordinate of the 1st node of the triangle
    __shared__ int node1_Y[MAX_TRIANGLE_COUNT_PER_BUCKET];  //!< Y coordinate of the 1st node of the triangle
    __shared__ int node1_Z[MAX_TRIANGLE_COUNT_PER_BUCKET];  //!< Z coordinate of the 1st node of the triangle

    __shared__ int node2_X[MAX_TRIANGLE_COUNT_PER_BUCKET];  //!< X coordinate of the 2nd node of the triangle
    __shared__ int node2_Y[MAX_TRIANGLE_COUNT_PER_BUCKET];  //!< Y coordinate of the 2nd node of the triangle
    __shared__ int node2_Z[MAX_TRIANGLE_COUNT_PER_BUCKET];  //!< Z coordinate of the 2nd node of the triangle

    __shared__ int node3_X[MAX_TRIANGLE_COUNT_PER_BUCKET];  //!< X coordinate of the 3rd node of the triangle
    __shared__ int node3_Y[MAX_TRIANGLE_COUNT_PER_BUCKET];  //!< Y coordinate of the 3rd node of the triangle
    __shared__ int node3_Z[MAX_TRIANGLE_COUNT_PER_BUCKET];  //!< Z coordinate of the 3rd node of the triangle

    volatile __shared__ float tempShMem[6 * (N_CUDATHREADS / warp_size)];  // used to do a block-level reduce

    float forceActingOnSphere[3];  //!< 3 registers will hold the value of the force on the sphere
    float genForceActingOnMeshes[TRIANGLE_FAMILIES * 6];  //!< 6 components per family: 3 forces and 3 torques

    unsigned int thisSD = blockIdx.x;
    unsigned int nSD_triangles = SD_countsOfTrianglesTouching[thisSD];
    unsigned int nSD_spheres = SD_countsOfGrElemsTouching[thisSD];

    if (nSD_triangles == 0 || nSD_spheres == 0)
        return;

    // Getting here means that there are both triangles and DEs in this SD.
    // First, figure out which bucket stores the triangles associated with this SD.
    unsigned int whichBKT = hashmapBKTid(thisSD) % TRIANGLEBUCKET_COUNT;
    unsigned int nBKT_triangles = BKT_countsOfTrianglesTouching[thisSD];

    // Unpleasant fact: this bucket might store more than the triangles associated with this SD. The narrow phase is
    // done for ALL triangles in this bucket with the expectation that if a triangle does not belong to this SD, the
    // narrow phase will prune this triangle fast and the penalty associated with storing triangles from multiple SDs
    // into one bucket is not stiff.

    // Populate the shared memory with terrain data
    unsigned int tripsToCoverSpheres = (nSD_spheres + blockDim.x - 1) / blockDim.x;
    unsigned int local_ID = threadIdx.x;
    for (unsigned int sphereTrip = 0; sphereTrip < tripsToCoverSpheres; sphereTrip++) {
        local_ID += sphereTrip * blockDim.x;
        if (local_ID < nSD_spheres) {
            unsigned int globalID = grElems_in_SD_composite[local_ID + thisSD * MAX_COUNT_OF_DEs_PER_SD];
            grElemID[local_ID] = globalID;
            sphX[local_ID] = d_sphere_pos_X[globalID];
            sphY[local_ID] = d_sphere_pos_Y[globalID];
            sphZ[local_ID] = d_sphere_pos_Z[globalID];
        }
    }
    // Populate the shared memory with mesh triangle data
    unsigned int tripsToCoverTriangles = (nBKT_triangles + blockDim.x - 1) / blockDim.x;
    local_ID = threadIdx.x;
    for (unsigned int triangTrip = 0; triangTrip < tripsToCoverTriangles; triangTrip++) {
        local_ID += triangTrip * blockDim.x;
        if (local_ID < nBKT_triangles) {
            unsigned int globalID = triangles_in_BKT_composite[local_ID + whichBKT * MAX_TRIANGLE_COUNT_PER_BUCKET];
            triangID[local_ID] = globalID;
            node1_X[local_ID] = d_triangleSoup->node1_X[globalID];
            node1_Y[local_ID] = d_triangleSoup->node1_Y[globalID];
            node1_Z[local_ID] = d_triangleSoup->node1_Z[globalID];

            node2_X[local_ID] = d_triangleSoup->node2_X[globalID];
            node2_Y[local_ID] = d_triangleSoup->node2_Y[globalID];
            node2_Z[local_ID] = d_triangleSoup->node2_Z[globalID];

            node3_X[local_ID] = d_triangleSoup->node3_X[globalID];
            node3_Y[local_ID] = d_triangleSoup->node3_Y[globalID];
            node3_Z[local_ID] = d_triangleSoup->node3_Z[globalID];
        }
    }

    __syncthreads();  // this call ensures data is in its place in shared memory

    /// Zero out the force and torque at the onset of the computation
    for (local_ID = 0; local_ID < TRIANGLE_FAMILIES; local_ID++) {
        unsigned int dummyOffset = 6 * local_ID;
        /// forces acting on the triangle, in global reference frame
        genForceActingOnMeshes[dummyOffset++] = 0.f;
        genForceActingOnMeshes[dummyOffset++] = 0.f;
        genForceActingOnMeshes[dummyOffset++] = 0.f;
        /// torques with respect to global reference frame, expressed in global reference frame
        genForceActingOnMeshes[dummyOffset++] = 0.f;
        genForceActingOnMeshes[dummyOffset++] = 0.f;
        genForceActingOnMeshes[dummyOffset++] = 0.f;
    }

    // Each sphere has one warp of threads dedicated to identifying all triangles that this sphere
    // touches. Upon a contact event, we'll compute the normal force on the sphere; and, the force and torque
    // impressed upon the triangle

    unsigned int nSpheresProcessedAtOneTime =
        blockDim.x / warp_size;  /// One warp allocated to slave serving one sphere
    tripsToCoverSpheres = (nSD_spheres + nSpheresProcessedAtOneTime - 1) / nSpheresProcessedAtOneTime;
    tripsToCoverTriangles = (nBKT_triangles + warp_size - 1) / warp_size;

    unsigned sphere_Local_ID = threadIdx.x / warp_size;
    for (unsigned int sphereTrip = 0; sphereTrip < tripsToCoverSpheres; sphereTrip++) {
        /// before starting dealing with a sphere, zero out the forces acting on it; all threads in the block are
        /// doing this
        forceActingOnSphere[0] = 0.f;
        forceActingOnSphere[1] = 0.f;
        forceActingOnSphere[2] = 0.f;
        sphere_Local_ID += sphereTrip * nSpheresProcessedAtOneTime;
        if (sphere_Local_ID < nSD_spheres) {
            /// Figure out which triangles this sphere collides with; each thread in a warp slaving for this sphere
            /// looks at one triangle at a time. The collection of threads in the warp sweeps through all the
            /// triangles that touch this SD. NOTE: to avoid double-counting, a sphere-triangle collision event is
            /// counted only if the collision point is in this SD.
            unsigned int targetTriangle = (threadIdx.x & (warp_size - 1));  // computes modulo 32 of the thread index
            for (unsigned int triangTrip = 0; triangTrip < tripsToCoverTriangles; triangTrip++) {
                targetTriangle += triangTrip * warp_size;
                if (targetTriangle < nBKT_triangles) {
                    /// we have a valid sphere and a valid triganle; check if in contact
                    double3 norm;
                    double depth;
                    double3 pt1;
                    double3 pt2;
                    double eff_radius;
                    double3 A = make_double3(node1_X[targetTriangle], node1_Y[targetTriangle], node1_Z[targetTriangle]);
                    double3 B = make_double3(node2_X[targetTriangle], node2_Y[targetTriangle], node2_Z[targetTriangle]);
                    double3 C = make_double3(node3_X[targetTriangle], node3_Y[targetTriangle], node3_Z[targetTriangle]);
                    double3 sphCntr = make_double3(sphX[sphere_Local_ID], sphY[sphere_Local_ID], sphZ[sphere_Local_ID]);
                    face_sphere_cd(A, B, C, sphCntr, gran_params->d_sphereRadius_SU, norm, depth, pt1, pt2, eff_radius);

                    /// Use the CD information to compute the force on the grElement

                    /// Use the CD information to compute the force and torque on the triangle
                }
            }
            /// down to the point where we need to collect the forces from all the threads in the wrap; this is a
            /// warp reduce operation. The resultant force acting on this grElement is stored in the first lane of
            /// the warp. NOTE: In this warp-level operations participate only the warps that are slaving for a
            /// sphere; i.e., some warps see no action
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[0] += __shfl_down_sync(0xffffffff, forceActingOnSphere[0], offset);
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[1] += __shfl_down_sync(0xffffffff, forceActingOnSphere[1], offset);
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[2] += __shfl_down_sync(0xffffffff, forceActingOnSphere[2], offset);

            /// done with the computation of all the contacts that the triangles impress on this sphere. Update the
            /// position of the sphere based on this force
        }
    }
    /// Done computing the forces acting on the triangles in this SD. A block reduce is carried out next. Start by
    /// doing a reduce at the warp level.
    for (local_ID = 0; local_ID < TRIANGLE_FAMILIES; local_ID++) {
        /// six generalized forces acting on the triangle, expressed in the global reference frame
        unsigned int dummyIndx = 6 * local_ID;
        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
        dummyIndx++;

        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
        dummyIndx++;

        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
        dummyIndx++;

        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
        dummyIndx++;

        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
        dummyIndx++;

        for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
            genForceActingOnMeshes[dummyIndx] +=
                __shfl_down_sync(0xffffffff, genForceActingOnMeshes[dummyIndx], offset);
    }

    __syncthreads();

    /// Lane zero in each warp holds the result of a warp-level reduce operation. Sum up these "Lane zero" values in
    /// the final result, which is block-level
    bool threadIsLaneZeroInWarp = ((threadIdx.x & (warp_size - 1)) == 0);
    for (local_ID = 0; local_ID < TRIANGLE_FAMILIES; local_ID++) {
        unsigned int offsetGenForceArray = 6 * local_ID;
        /// Place in ShMem forces/torques (expressed in global reference frame) acting on this family of triangles
        if (threadIsLaneZeroInWarp) {
            unsigned int offsetShMem = 6 * (threadIdx.x / warp_size);
            tempShMem[offsetShMem++] = genForceActingOnMeshes[offsetGenForceArray++];
            tempShMem[offsetShMem++] = genForceActingOnMeshes[offsetGenForceArray++];
            tempShMem[offsetShMem++] = genForceActingOnMeshes[offsetGenForceArray++];

            tempShMem[offsetShMem++] = genForceActingOnMeshes[offsetGenForceArray++];
            tempShMem[offsetShMem++] = genForceActingOnMeshes[offsetGenForceArray++];
            tempShMem[offsetShMem] = genForceActingOnMeshes[offsetGenForceArray];
        }
        __syncthreads();

        /// Going to trash the values in "forceActingOnSphere", not needed anymore. Reuse the registers, which will
        /// now store the vaule of the triangle force and torque...
        if (threadIdx.x < warp_size) {
            /// only first thread in block participates in this reduce operation.
            /// NOTE: an implicit assumption is made here - warp_size is larger than or equal to N_CUDATHREADS /
            /// warp_size. This is true today as N_CUDATHREADS cannot be larger than 1024 and warp_size is 32.

            /// Work on forces first. Place data from ShMem into registers associated w/ first warp
            unsigned int offsetShMem = 6 * threadIdx.x;
            if (threadIdx.x < (N_CUDATHREADS / warp_size)) {
                forceActingOnSphere[0] = tempShMem[offsetShMem++];
                forceActingOnSphere[1] = tempShMem[offsetShMem++];
                forceActingOnSphere[2] = tempShMem[offsetShMem++];
            } else {
                /// this is hit only by a subset of threads from first warp of the block
                forceActingOnSphere[0] = 0.f;
                forceActingOnSphere[1] = 0.f;
                forceActingOnSphere[2] = 0.f;
            }

            offsetGenForceArray = 6 * local_ID;
            // X component of the force on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[0] += __shfl_down_sync(0xffffffff, forceActingOnSphere[0], offset);
            genForceActingOnMeshes[offsetGenForceArray++] = forceActingOnSphere[0];

            // Y component of the force on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[1] += __shfl_down_sync(0xffffffff, forceActingOnSphere[1], offset);
            genForceActingOnMeshes[offsetGenForceArray++] = forceActingOnSphere[1];

            // Z component of the force on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[2] += __shfl_down_sync(0xffffffff, forceActingOnSphere[2], offset);
            genForceActingOnMeshes[offsetGenForceArray++] = forceActingOnSphere[2];

            /// Finally, work on torques
            if (threadIdx.x < (N_CUDATHREADS / warp_size)) {
                forceActingOnSphere[0] = tempShMem[offsetShMem++];
                forceActingOnSphere[1] = tempShMem[offsetShMem++];
                forceActingOnSphere[2] = tempShMem[offsetShMem];
            } else {
                /// this is hit only by a subset of threads from first warp of the block
                forceActingOnSphere[0] = 0.f;
                forceActingOnSphere[1] = 0.f;
                forceActingOnSphere[2] = 0.f;
            }

            // X component of the torque on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[0] += __shfl_down_sync(0xffffffff, forceActingOnSphere[0], offset);
            genForceActingOnMeshes[offsetGenForceArray++] = forceActingOnSphere[0];

            // Y component of the torque on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[1] += __shfl_down_sync(0xffffffff, forceActingOnSphere[1], offset);
            genForceActingOnMeshes[offsetGenForceArray++] = forceActingOnSphere[1];

            // Z component of the torque on mesh "local_ID"
            for (unsigned int offset = warp_size / 2; offset > 0; offset /= 2)
                forceActingOnSphere[2] += __shfl_down_sync(0xffffffff, forceActingOnSphere[2], offset);
            genForceActingOnMeshes[offsetGenForceArray] = forceActingOnSphere[2];
        }  /// this is the end of the "for each mesh" loop

        /// At this point, the first thread of the block has in genForceActingOnMeshes[6*TRIANGLE_FAMILIES] the
        /// forces and torques acting on each mesh family. Bcast the force values to all threads in the warp.
        /// To this end, synchronize all threads in warp and get "value" from lane 0
        for (local_ID = 0; local_ID < 6 * TRIANGLE_FAMILIES; local_ID++)
            genForceActingOnMeshes[local_ID] = __shfl_sync(0xffffffff, genForceActingOnMeshes[local_ID], 0);

        /// At this point, all threads in the first warp have the generalized forces acting on all meshes. Do an
        /// atomic add to compund the value of the generalized forces acting on the meshes that come in contact with
        /// the granular material.
        unsigned int nTrips = (6 * TRIANGLE_FAMILIES) / warp_size;
        for (local_ID = 0; local_ID < nTrips + 1; local_ID++) {
            unsigned int offset = threadIdx.x + local_ID * (6 * TRIANGLE_FAMILIES);
            if (offset < 6 * TRIANGLE_FAMILIES)
                atomicAdd(d_triangleSoup->generalizedForcesPerFamily + offset, genForceActingOnMeshes[offset]);
        }
    }
}

/// Copy const triangle data to device
void chrono::granular::ChSystemGranularMonodisperse_SMC_Frictionless_trimesh::copy_triangle_data_to_device() {
    // unified memory does some copying for us, cool
    tri_params->d_Gamma_n_s2m_SU = 0;  // no damping on mesh for now
    tri_params->d_Kn_s2m_SU = 7;       // TODO Nic you get to deal with this
    // tri_params->num_triangle_families = 4;  // TODO make this legit
    // Or Conlain can deal with it later, no way this actually runs cleanly anyways
}

__host__ void chrono::granular::ChSystemGranularMonodisperse_SMC_Frictionless_trimesh::initialize() {
    switch_to_SimUnits();
    generate_DEs();

    // Set aside memory for holding data structures worked with. Get some initializations going
    setup_simulation();
    copy_const_data_to_device();
    copy_triangle_data_to_device();
    copyBD_Frame_to_device();
    gpuErrchk(hipDeviceSynchronize());

    // Seed arrays that are populated by the kernel call
    resetBroadphaseInformation();

    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nDEs + CUDA_THREADS - 1) / CUDA_THREADS;
    printf("doing priming!\n");
    printf("max possible composite offset is %zu\n", (size_t)nSDs * MAX_COUNT_OF_DEs_PER_SD);

    primingOperationsRectangularBox<CUDA_THREADS>
        <<<nBlocks, CUDA_THREADS>>>(pos_X.data(), pos_Y.data(), pos_Z.data(), SD_NumOf_DEs_Touching.data(),
                                    DEs_in_SD_composite.data(), nDEs, gran_params);
    gpuErrchk(hipDeviceSynchronize());
    printf("priming finished!\n");

    VERBOSE_PRINTF("z grav term with timestep %u is %f\n", stepSize_SU, stepSize_SU * stepSize_SU * gravity_Z_SU);
}

__host__ void chrono::granular::ChSystemGranularMonodisperse_SMC_Frictionless_trimesh::advance_simulation(
    float duration) {
    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nDEs + CUDA_THREADS - 1) / CUDA_THREADS;

    // Settling simulation loop.
    unsigned int duration_SU = std::ceil(duration / (TIME_UNIT * PSI_h));
    unsigned int nsteps = (1.0 * duration_SU) / stepSize_SU;

    printf("advancing by %u at timestep %u, %u timesteps at approx user timestep %f\n", duration_SU, stepSize_SU,
           nsteps, duration / nsteps);

    VERBOSE_PRINTF("Starting Main Simulation loop!\n");
    // Run the simulation, there are aggressive synchronizations because we want to have no race conditions
    for (unsigned int crntTime_SU = 0; crntTime_SU < stepSize_SU * nsteps; crntTime_SU += stepSize_SU) {
        // Update the position and velocity of the BD, if relevant
        if (!BD_is_fixed) {
            updateBDPosition(stepSize_SU);
        }
        resetUpdateInformation();
        update_DMeshSoup_Location();  // TODO where does this go?

        VERBOSE_PRINTF("Starting computeVelocityUpdates!\n");

        // Compute forces and crank into vel updates, we have 2 kernels to avoid a race condition
        computeVelocityUpdates<MAX_COUNT_OF_DEs_PER_SD><<<nSDs, MAX_TRIANGLE_COUNT_PER_BUCKET>>>(
            stepSize_SU, pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt_update.data(), pos_Y_dt_update.data(),
            pos_Z_dt_update.data(), SD_NumOf_DEs_Touching.data(), DEs_in_SD_composite.data(), pos_X_dt.data(),
            pos_Y_dt.data(), pos_Z_dt.data(), gran_params);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // broadphase the triangles
        // todo teh mesh soup needs to be unified memory I think
        triangleSoupBroadPhase<CUDA_THREADS><<<nSDs, MAX_COUNT_OF_DEs_PER_SD>>>(
            meshSoup_DEVICE, BUCKET_countsOfTrianglesTouching.data(), triangles_in_BUCKET_composite.data(),
            SD_countsOfTrianglesTouching.data(), gran_params, tri_params);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        // TODO please do not use a template here
        // compute sphere-triangle forces
        interactionTerrain_TriangleSoup<CUDA_THREADS><<<nSDs, MAX_COUNT_OF_DEs_PER_SD>>>(
            meshSoup_DEVICE, pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt_update.data(), pos_Y_dt_update.data(),
            pos_Z_dt_update.data(), SD_NumOf_DEs_Touching.data(), DEs_in_SD_composite.data(),
            BUCKET_countsOfTrianglesTouching.data(), triangles_in_BUCKET_composite.data(),
            SD_countsOfTrianglesTouching.data(), gran_params, tri_params);

        // gpuErrchk(hipPeekAtLastError());
        // gpuErrchk(hipDeviceSynchronize());
        //
        // VERBOSE_PRINTF("Starting applyVelocityUpdates!\n");
        // // Apply the updates we just made
        // applyVelocityUpdates<MAX_COUNT_OF_DEs_PER_SD><<<nSDs, MAX_COUNT_OF_DEs_PER_SD>>>(
        //     stepSize_SU, pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt_update.data(), pos_Y_dt_update.data(),
        //     pos_Z_dt_update.data(), SD_NumOf_DEs_Touching.data(), DEs_in_SD_composite.data(), pos_X_dt.data(),
        //     pos_Y_dt.data(), pos_Z_dt.data(), gran_params);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        VERBOSE_PRINTF("Resetting broadphase info!\n");

        resetBroadphaseInformation();

        VERBOSE_PRINTF("Starting updatePositions!\n");
        updatePositions<CUDA_THREADS><<<nBlocks, CUDA_THREADS>>>(
            stepSize_SU, pos_X.data(), pos_Y.data(), pos_Z.data(), pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data(),
            pos_X_dt_update.data(), pos_Y_dt_update.data(), pos_Z_dt_update.data(), SD_NumOf_DEs_Touching.data(),
            DEs_in_SD_composite.data(), nDEs, gran_params);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }
    return;
}
